#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

/*
 * A simple example of nested kernel launches from the GPU. Each thread displays
 * its information when execution begins, and also diagnostics when the next
 * lowest nesting layer completes.
 */

__global__ void nestedHelloWorld(int const iSize, int iDepth, int maxDepth)
{
    int tid = threadIdx.x;
    printf("Recursion=%d: Hello World from thread %d block %d\n", iDepth, tid,
           blockIdx.x);

    // condition to stop recursive execution
    if (iSize == 1 || iDepth >= maxDepth) return;

    // reduce block size to half
    int nthreads = iSize >> 1;

    // thread 0 launches child grid recursively
    if(tid == 0 && nthreads > 0)
    {
        nestedHelloWorld<<<1, nthreads>>>(nthreads, ++iDepth, maxDepth);
        printf("-------> nested execution depth: %d\n", iDepth);
    }
}

int main(int argc, char **argv)
{
    int size = 8;
    int blocksize = 8;   // initial block size
    int igrid = 1;

    if(argc > 1)
    {
        igrid = atoi(argv[1]);
        size = igrid * blocksize;
    }

    dim3 block (blocksize, 1);
    dim3 grid  ((size + block.x - 1) / block.x, 1);
    printf("%s Execution Configuration: grid %d block %d\n", argv[0], grid.x,
           block.x);

    nestedHelloWorld<<<grid, block>>>(block.x, 0, 2);

    CHECK(hipGetLastError());
    CHECK(hipDeviceReset());
    return 0;
}
