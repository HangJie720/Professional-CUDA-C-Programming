#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * This code implements the interleaved and neighbor-paired approaches to
 * parallel reduction in CUDA. For this example, the sum operation is used. A
 * variety of optimizations on parallel reduction aimed at reducing divergence
 * are also demonstrated, such as unrolling.
 */

// Recursive Implementation of Interleaved Pair Approach
int recursiveReduce(int *data, int const size)
{
    // terminate check
    if (size == 1) return data[0];

    // renew the stride
    int const stride = size / 2;

    // in-place reduction
    for (int i = 0; i < stride; i++)
    {
        data[i] += data[i + stride];
    }

    // call recursively
    return recursiveReduce(data, stride);
}

__global__ void warmup(int *out, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N)
    {
        out[tid] = 0;
    }
}

__global__ void reduceCompleteUnrollWarps8 (int *g_idata, int *g_odata,
        unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // unrolling 8
    if (idx + 7 * blockDim.x < n)
    {
        int a1 = g_idata[idx];
        int a2 = g_idata[idx + blockDim.x];
        int a3 = g_idata[idx + 2 * blockDim.x];
        int a4 = g_idata[idx + 3 * blockDim.x];
        int b1 = g_idata[idx + 4 * blockDim.x];
        int b2 = g_idata[idx + 5 * blockDim.x];
        int b3 = g_idata[idx + 6 * blockDim.x];
        int b4 = g_idata[idx + 7 * blockDim.x];
        g_idata[idx] = a1 + a2 + a3 + a4 + b1 + b2 + b3 + b4;
    }

    __syncthreads();

    // in-place reduction and complete unroll
    if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];

    __syncthreads();

    if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];

    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];

    __syncthreads();

    if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];

    __syncthreads();

    // unrolling warp
    if (tid < 32)
    {
        volatile int *vmem = idata;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid +  8];
        vmem[tid] += vmem[tid +  4];
        vmem[tid] += vmem[tid +  2];
        vmem[tid] += vmem[tid +  1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceCompleteUnrollWarps8Sync(int *g_idata, int *g_odata,
        unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // unrolling 8
    if (idx + 7 * blockDim.x < n)
    {
        int a1 = g_idata[idx];
        int a2 = g_idata[idx + blockDim.x];
        int a3 = g_idata[idx + 2 * blockDim.x];
        int a4 = g_idata[idx + 3 * blockDim.x];
        int b1 = g_idata[idx + 4 * blockDim.x];
        int b2 = g_idata[idx + 5 * blockDim.x];
        int b3 = g_idata[idx + 6 * blockDim.x];
        int b4 = g_idata[idx + 7 * blockDim.x];
        g_idata[idx] = a1 + a2 + a3 + a4 + b1 + b2 + b3 + b4;
    }

    __syncthreads();

    // in-place reduction and complete unroll
    if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];

    __syncthreads();

    if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];

    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];

    __syncthreads();

    if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];

    __syncthreads();

    // unrolling warp
    if (tid < 32)
        idata[tid] += idata[tid + 32];
    __syncthreads();

    if (tid < 32)
        idata[tid] += idata[tid + 16];
    __syncthreads();

    if (tid < 32)
        idata[tid] += idata[tid +  8];
    __syncthreads();

    if (tid < 32)
        idata[tid] += idata[tid +  4];
    __syncthreads();

    if (tid < 32)
        idata[tid] += idata[tid +  2];
    __syncthreads();

    if (tid < 32)
        idata[tid] += idata[tid +  1];

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceCompleteUnrollWarps8SyncVolatile(int *g_idata, int *g_odata,
        unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // unrolling 8
    if (idx + 7 * blockDim.x < n)
    {
        int a1 = g_idata[idx];
        int a2 = g_idata[idx + blockDim.x];
        int a3 = g_idata[idx + 2 * blockDim.x];
        int a4 = g_idata[idx + 3 * blockDim.x];
        int b1 = g_idata[idx + 4 * blockDim.x];
        int b2 = g_idata[idx + 5 * blockDim.x];
        int b3 = g_idata[idx + 6 * blockDim.x];
        int b4 = g_idata[idx + 7 * blockDim.x];
        g_idata[idx] = a1 + a2 + a3 + a4 + b1 + b2 + b3 + b4;
    }

    __syncthreads();

    // in-place reduction and complete unroll
    if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];

    __syncthreads();

    if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];

    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];

    __syncthreads();

    if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];

    __syncthreads();

    volatile int *vmem = idata;
    // unrolling warp
    if (tid < 32)
        idata[tid] += vmem[tid + 32];
    __syncthreads();

    if (tid < 32)
        idata[tid] += vmem[tid + 16];
    __syncthreads();

    if (tid < 32)
        idata[tid] += vmem[tid +  8];
    __syncthreads();

    if (tid < 32)
        idata[tid] += vmem[tid +  4];
    __syncthreads();

    if (tid < 32)
        idata[tid] += vmem[tid +  2];
    __syncthreads();

    if (tid < 32)
        idata[tid] += vmem[tid +  1];

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s starting reduction at ", argv[0]);
    printf("device %d: %s ", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    bool bResult = false;

    // initialization
    int size = 1 << 24; // total number of elements to reduce
    printf("    with array size %d  ", size);

    // execution configuration
    int blocksize = 512;   // initial block size

    if(argc > 1)
    {
        blocksize = atoi(argv[1]);   // block size from command line argument
    }

    dim3 block (blocksize, 1);
    dim3 grid  ((size + block.x - 1) / block.x, 1);
    printf("grid %d block %d\n", grid.x, block.x);

    // allocate host memory
    size_t bytes = size * sizeof(int);
    int *h_idata = (int *) malloc(bytes);
    int *h_odata = (int *) malloc(grid.x * sizeof(int));
    int *tmp     = (int *) malloc(bytes);

    // initialize the array
    for (int i = 0; i < size; i++)
    {
        // mask off high 2 bytes to force max number to 255
        h_idata[i] = (int)( rand() & 0xFF );
    }

    memcpy (tmp, h_idata, bytes);

    double iStart, iElaps;
    int gpu_sum = 0;

    // allocate device memory
    int *d_idata = NULL;
    int *d_odata = NULL;
    CHECK(hipMalloc((void **) &d_idata, bytes));
    CHECK(hipMalloc((void **) &d_odata, grid.x * sizeof(int)));

    // cpu reduction
    iStart = seconds();
    int cpu_sum = recursiveReduce (tmp, size);
    iElaps = seconds() - iStart;
    printf("cpu reduce      elapsed %f sec cpu_sum: %d\n", iElaps, cpu_sum);

    // Warmup
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    iStart = seconds();
    warmup<<<grid.x, block>>>(d_odata, grid.x);
    CHECK(hipDeviceSynchronize());

    // No sync
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    iStart = seconds();
    reduceCompleteUnrollWarps8<<<grid.x / 8, block>>>(d_idata, d_odata, size);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    CHECK(hipMemcpy(h_odata, d_odata, grid.x / 8 * sizeof(int),
                     hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x / 8; i++) gpu_sum += h_odata[i];

    printf("gpu Cmptnroll8  elapsed %f sec gpu_sum: %d <<<grid %d block "
           "%d>>>\n", iElaps, gpu_sum, grid.x / 8, block.x);

    // Sync
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    iStart = seconds();
    reduceCompleteUnrollWarps8Sync<<<grid.x / 8, block>>>(d_idata, d_odata,
            size);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    CHECK(hipMemcpy(h_odata, d_odata, grid.x / 8 * sizeof(int),
                     hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x / 8; i++) gpu_sum += h_odata[i];

    printf("gpu Cmptnroll8Sync  elapsed %f sec gpu_sum: %d <<<grid %d block "
           "%d>>>\n", iElaps, gpu_sum, grid.x / 8, block.x);

    // Sync with Volatile
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    iStart = seconds();
    reduceCompleteUnrollWarps8SyncVolatile<<<grid.x / 8, block>>>(d_idata, d_odata,
            size);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    CHECK(hipMemcpy(h_odata, d_odata, grid.x / 8 * sizeof(int),
                     hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x / 8; i++) gpu_sum += h_odata[i];

    printf("gpu Cmptnroll8SyncVol  elapsed %f sec gpu_sum: %d <<<grid %d block "
           "%d>>>\n", iElaps, gpu_sum, grid.x / 8, block.x);


    // free host memory
    free(h_idata);
    free(h_odata);

    // free device memory
    CHECK(hipFree(d_idata));
    CHECK(hipFree(d_odata));

    // reset device
    CHECK(hipDeviceReset());

    // check the results
    bResult = (gpu_sum == cpu_sum);

    if(!bResult) printf("Test failed!\n");

    return EXIT_SUCCESS;
}
