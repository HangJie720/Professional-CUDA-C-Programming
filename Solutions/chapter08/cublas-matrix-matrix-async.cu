#include "../common/common.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

/*
 * A simple example of performing matrix-vector multiplication using the cuBLAS
 * library and some randomly generated inputs.
 */

/*
 * M = # of rows
 * N = # of columns
 */
int M = 1024;
int N = 1024;

/*
 * Generate a matrix with M rows and N columns in column-major order. The matrix
 * will be filled with random single-precision floating-point values between 0
 * and 100.
 */
void generate_random_dense_matrix(int M, int N, float **outA)
{
    int i, j;
    double rMax = (double)RAND_MAX;
    float *A = (float *)malloc(sizeof(float) * M * N);

    // For each column
    for (j = 0; j < N; j++)
    {
        // For each row
        for (i = 0; i < M; i++)
        {
            double dr = (double)rand();
            A[j * M + i] = (dr / rMax) * 100.0;
        }
    }

    *outA = A;
}

int main(int argc, char **argv)
{
    int i, j;
    float *A, *dA;
    float *B, *dB;
    float *C, *dC;
    float beta;
    float alpha;
    hipblasHandle_t handle = 0;
    hipStream_t stream = 0;

    alpha = 3.0f;
    beta = 4.0f;

    // Generate inputs
    srand(9384);
    generate_random_dense_matrix(M, N, &A);
    generate_random_dense_matrix(N, M, &B);
    C = (float *)malloc(sizeof(float) * M * M);
    memset(C, 0x00, sizeof(float) * M * M);

    // Create the cuBLAS handle
    CHECK_CUBLAS(hipblasCreate(&handle));
    CHECK(hipStreamCreate(&stream));
    CHECK_CUBLAS(hipblasSetStream(handle, stream));

    // Allocate device memory
    CHECK(hipMalloc((void **)&dA, sizeof(float) * M * N));
    CHECK(hipMalloc((void **)&dB, sizeof(float) * N * M));
    CHECK(hipMalloc((void **)&dC, sizeof(float) * M * M));

    // Transfer inputs to the device
    CHECK_CUBLAS(hipblasSetMatrixAsync(M, N, sizeof(float), A, M, dA, M,
                stream));
    CHECK_CUBLAS(hipblasSetMatrixAsync(N, M, sizeof(float), B, N, dB, N,
                stream));
    CHECK_CUBLAS(hipblasSetMatrixAsync(M, M, sizeof(float), C, M, dC, M,
                stream));

    // Execute the matrix-vector multiplication
    CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, M, N, &alpha,
                dA, M, dB, N, &beta, dC, M));

    // Retrieve the output vector from the device
    CHECK_CUBLAS(hipblasGetMatrixAsync(M, M, sizeof(float), dC, M, C, M,
                stream));
    CHECK(hipStreamSynchronize(stream));

    for (j = 0; j < 10; j++)
    {
        for (i = 0; i < 10; i++)
        {
            printf("%2.2f ", C[j * M + i]);
        }
        printf("...\n");
    }

    printf("...\n");

    free(A);
    free(B);
    free(C);

    CHECK(hipFree(dA));
    CHECK(hipFree(dB));
    CHECK(hipFree(dC));
    CHECK(hipStreamDestroy(stream));
    CHECK_CUBLAS(hipblasDestroy(handle));

    return 0;
}
