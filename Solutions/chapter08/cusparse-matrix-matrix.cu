#include "../common/common.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>

/*
 * This is an example demonstrating usage of the cuSPARSE library to perform a
 * sparse matrix-vector multiplication on randomly generated data.
 */

/*
 * M = # of rows
 * N = # of columns
 */
int M = 1024;
int N = 1024;

/*
 * Generate random dense matrix A in column-major order, while rounding some
 * elements down to zero to ensure it is sparse.
 */
int generate_random_dense_matrix(int M, int N, float **outA)
{
    int i, j;
    double rMax = (double)RAND_MAX;
    float *A = (float *)malloc(sizeof(float) * M * N);
    int totalNnz = 0;

    for (j = 0; j < N; j++)
    {
        for (i = 0; i < M; i++)
        {
            int r = rand();
            float *curr = A + (j * M + i);

            if (r % 3 > 0)
            {
                *curr = 0.0f;
            }
            else
            {
                double dr = (double)r;
                *curr = (dr / rMax) * 100.0;
            }

            if (*curr != 0.0f)
            {
                totalNnz++;
            }
        }
    }

    *outA = A;
    return totalNnz;
}

void print_partial_matrix(float *M, int nrows, int ncols, int max_row,
        int max_col)
{
    int row, col;

    for (row = 0; row < max_row; row++)
    {
        for (col = 0; col < max_col; col++)
        {
            printf("%2.2f ", M[row * ncols + col]);
        }
        printf("...\n");
    }
    printf("...\n");
}

int main(int argc, char **argv)
{
    float *A, *dA;
    float *B, *dB;
    float *C, *dC;
    int *dANnzPerRow;
    float *dCsrValA;
    int *dCsrRowPtrA;
    int *dCsrColIndA;
    int totalANnz;
    float alpha = 3.0f;
    float beta = 4.0f;
    hipsparseHandle_t handle = 0;
    hipsparseMatDescr_t Adescr = 0;

    // Generate input
    srand(9384);
    int trueANnz = generate_random_dense_matrix(M, N, &A);
    int trueBNnz = generate_random_dense_matrix(N, M, &B);
    C = (float *)malloc(sizeof(float) * M * M);

    printf("A:\n");
    print_partial_matrix(A, M, N, 10, 10);
    printf("B:\n");
    print_partial_matrix(B, N, M, 10, 10);

    // Create the cuSPARSE handle
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    // Allocate device memory for vectors and the dense form of the matrix A
    CHECK(hipMalloc((void **)&dA, sizeof(float) * M * N));
    CHECK(hipMalloc((void **)&dB, sizeof(float) * N * M));
    CHECK(hipMalloc((void **)&dC, sizeof(float) * M * M));
    CHECK(hipMalloc((void **)&dANnzPerRow, sizeof(int) * M));

    // Construct a descriptor of the matrix A
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&Adescr));
    CHECK_CUSPARSE(hipsparseSetMatType(Adescr, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(Adescr, HIPSPARSE_INDEX_BASE_ZERO));

    // Transfer the input vectors and dense matrix A to the device
    CHECK(hipMemcpy(dA, A, sizeof(float) * M * N, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dB, B, sizeof(float) * N * M, hipMemcpyHostToDevice));
    CHECK(hipMemset(dC, 0x00, sizeof(float) * M * M));

    // Compute the number of non-zero elements in A
    CHECK_CUSPARSE(hipsparseSnnz(handle, HIPSPARSE_DIRECTION_ROW, M, N, Adescr,
                                dA, M, dANnzPerRow, &totalANnz));

    if (totalANnz != trueANnz)
    {
        fprintf(stderr, "Difference detected between cuSPARSE NNZ and true "
                "value: expected %d but got %d\n", trueANnz, totalANnz);
        return 1;
    }

    // Allocate device memory to store the sparse CSR representation of A
    CHECK(hipMalloc((void **)&dCsrValA, sizeof(float) * totalANnz));
    CHECK(hipMalloc((void **)&dCsrRowPtrA, sizeof(int) * (M + 1)));
    CHECK(hipMalloc((void **)&dCsrColIndA, sizeof(int) * totalANnz));

    // Convert A from a dense formatting to a CSR formatting, using the GPU
    CHECK_CUSPARSE(hipsparseSdense2csr(handle, M, N, Adescr, dA, M, dANnzPerRow,
                                      dCsrValA, dCsrRowPtrA, dCsrColIndA));

    // Perform matrix-matrix multiplication with the CSR-formatted matrix A
    CHECK_CUSPARSE(hipsparseScsrmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, M,
                                  M, N, totalANnz, &alpha, Adescr, dCsrValA,
                                  dCsrRowPtrA, dCsrColIndA, dB, N, &beta, dC,
                                  M));

    // Copy the result vector back to the host
    CHECK(hipMemcpy(C, dC, sizeof(float) * M * M, hipMemcpyDeviceToHost));

    printf("C:\n");
    print_partial_matrix(C, M, M, 10, 10);

    free(A);
    free(B);
    free(C);

    CHECK(hipFree(dA));
    CHECK(hipFree(dB));
    CHECK(hipFree(dC));
    CHECK(hipFree(dANnzPerRow));
    CHECK(hipFree(dCsrValA));
    CHECK(hipFree(dCsrRowPtrA));
    CHECK(hipFree(dCsrColIndA));

    CHECK_CUSPARSE(hipsparseDestroyMatDescr(Adescr));
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return 0;
}
