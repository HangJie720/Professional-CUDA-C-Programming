#include "../common/common.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>

/*
 * This is an example demonstrating usage of the cuSPARSE library to perform a
 * sparse matrix-vector multiplication on randomly generated data.
 */

/*
 * M = # of rows
 * N = # of columns
 */
int M = 1024;
int N = 1024;

/*
 * Generate random dense matrix A in column-major order, while rounding some
 * elements down to zero to ensure it is sparse.
 */
int generate_random_dense_matrix(int M, int N, double **outA)
{
    int i, j;
    double rMax = (double)RAND_MAX;
    double *A = (double *)malloc(sizeof(double) * M * N);
    int totalNnz = 0;

    for (j = 0; j < N; j++)
    {
        for (i = 0; i < M; i++)
        {
            int r = rand();
            double *curr = A + (j * M + i);

            if (r % 3 > 0)
            {
                *curr = 0.0;
            }
            else
            {
                double dr = (double)r;
                *curr = (dr / rMax) * 100.0;
            }

            if (*curr != 0.0)
            {
                totalNnz++;
            }
        }
    }

    *outA = A;
    return totalNnz;
}

void print_partial_matrix(double *M, int nrows, int ncols, int max_row,
        int max_col)
{
    int row, col;

    for (row = 0; row < max_row; row++)
    {
        for (col = 0; col < max_col; col++)
        {
            printf("%2.2f ", M[row * ncols + col]);
        }
        printf("...\n");
    }
    printf("...\n");
}

int main(int argc, char **argv)
{
    double *A, *dA;
    double *B, *dB;
    double *C, *dC;
    int *dANnzPerRow;
    double *dCsrValA;
    int *dCsrRowPtrA;
    int *dCsrColIndA;
    int totalANnz;
    double alpha = 3.0;
    double beta = 4.0;
    hipsparseHandle_t handle = 0;
    hipsparseMatDescr_t Adescr = 0;

    // Generate input
    srand(9384);
    int trueANnz = generate_random_dense_matrix(M, N, &A);
    int trueBNnz = generate_random_dense_matrix(N, M, &B);
    C = (double *)malloc(sizeof(double) * M * M);

    printf("A:\n");
    print_partial_matrix(A, M, N, 10, 10);
    printf("B:\n");
    print_partial_matrix(B, N, M, 10, 10);

    // Create the cuSPARSE handle
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    // Allocate device memory for vectors and the dense form of the matrix A
    CHECK(hipMalloc((void **)&dA, sizeof(double) * M * N));
    CHECK(hipMalloc((void **)&dB, sizeof(double) * N * M));
    CHECK(hipMalloc((void **)&dC, sizeof(double) * M * M));
    CHECK(hipMalloc((void **)&dANnzPerRow, sizeof(int) * M));

    // Construct a descriptor of the matrix A
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&Adescr));
    CHECK_CUSPARSE(hipsparseSetMatType(Adescr, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(Adescr, HIPSPARSE_INDEX_BASE_ZERO));

    // Transfer the input vectors and dense matrix A to the device
    CHECK(hipMemcpy(dA, A, sizeof(double) * M * N, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dB, B, sizeof(double) * N * M, hipMemcpyHostToDevice));
    CHECK(hipMemset(dC, 0x00, sizeof(double) * M * M));

    // Compute the number of non-zero elements in A
    CHECK_CUSPARSE(hipsparseDnnz(handle, HIPSPARSE_DIRECTION_ROW, M, N, Adescr,
                                dA, M, dANnzPerRow, &totalANnz));

    if (totalANnz != trueANnz)
    {
        fprintf(stderr, "Difference detected between cuSPARSE NNZ and true "
                "value: expected %d but got %d\n", trueANnz, totalANnz);
        return 1;
    }

    // Allocate device memory to store the sparse CSR representation of A
    CHECK(hipMalloc((void **)&dCsrValA, sizeof(double) * totalANnz));
    CHECK(hipMalloc((void **)&dCsrRowPtrA, sizeof(int) * (M + 1)));
    CHECK(hipMalloc((void **)&dCsrColIndA, sizeof(int) * totalANnz));

    // Convert A from a dense formatting to a CSR formatting, using the GPU
    CHECK_CUSPARSE(hipsparseDdense2csr(handle, M, N, Adescr, dA, M, dANnzPerRow,
                                      dCsrValA, dCsrRowPtrA, dCsrColIndA));

    // Perform matrix-matrix multiplication with the CSR-formatted matrix A
    CHECK_CUSPARSE(hipsparseDcsrmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, M,
                                  M, N, totalANnz, &alpha, Adescr, dCsrValA,
                                  dCsrRowPtrA, dCsrColIndA, dB, N, &beta, dC,
                                  M));

    // Copy the result vector back to the host
    CHECK(hipMemcpy(C, dC, sizeof(double) * M * M, hipMemcpyDeviceToHost));

    printf("C:\n");
    print_partial_matrix(C, M, M, 10, 10);

    free(A);
    free(B);
    free(C);

    CHECK(hipFree(dA));
    CHECK(hipFree(dB));
    CHECK(hipFree(dC));
    CHECK(hipFree(dANnzPerRow));
    CHECK(hipFree(dCsrValA));
    CHECK(hipFree(dCsrRowPtrA));
    CHECK(hipFree(dCsrColIndA));

    CHECK_CUSPARSE(hipsparseDestroyMatDescr(Adescr));
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return 0;
}
