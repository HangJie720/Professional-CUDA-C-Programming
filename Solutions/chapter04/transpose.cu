#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Various memory access pattern optimizations applied to a matrix transpose
 * kernel.
 */

#define BDIMX 16
#define BDIMY 16

void initialData(float *in,  const int size)
{
    for (int i = 0; i < size; i++)
    {
        in[i] = (float)( rand() & 0xFF ) / 10.0f; //100.0f;
    }

    return;
}

void printData(float *in,  const int size)
{
    for (int i = 0; i < size; i++)
    {
        printf("%dth element: %f\n", i, in[i]);
    }

    return;
}

void checkResult(float *hostRef, float *gpuRef, const int size, int showme)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < size; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("different on %dth element: host %f gpu %f\n", i, hostRef[i],
                    gpuRef[i]);
            break;
        }

        if (showme && i > size / 2 && i < size / 2 + 5)
        {
            // printf("%dth element: host %f gpu %f\n",i,hostRef[i],gpuRef[i]);
        }
    }

    if (!match)  printf("Arrays do not match.\n\n");
}

void transposeHost(float *out, float *in, const int nx, const int ny)
{
    for( int iy = 0; iy < ny; ++iy)
    {
        for( int ix = 0; ix < nx; ++ix)
        {
            out[ix * ny + iy] = in[iy * nx + ix];
        }
    }
}

__global__ void warmup(float *out, float *in, const int nx, const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix < nx && iy < ny)
    {
        out[iy * nx + ix] = in[iy * nx + ix];
    }
}

// case 0 copy kernel: access data in rows
__global__ void copyRow(float *out, float *in, const int nx, const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix < nx && iy < ny)
    {
        out[iy * nx + ix] = in[iy * nx + ix];
    }
}

// case 1 copy kernel: access data in columns
__global__ void copyCol(float *out, float *in, const int nx, const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix < nx && iy < ny)
    {
        out[ix * ny + iy] = in[ix * ny + iy];
    }
}

// case 2 transpose kernel: read in rows and write in columns
__global__ void transposeNaiveRow(float *out, float *in, const int nx,
                                  const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix < nx && iy < ny)
    {
        out[ix * ny + iy] = in[iy * nx + ix];
    }
}

// case 3 transpose kernel: read in columns and write in rows
__global__ void transposeNaiveCol(float *out, float *in, const int nx,
                                  const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix < nx && iy < ny)
    {
        out[iy * nx + ix] = in[ix * ny + iy];
    }
}

// case 4 transpose kernel: read in rows and write in columns + unroll 4 blocks
__global__ void transposeUnroll4Row(float *out, float *in, const int nx,
                                    const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x * 4 + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    unsigned int ti = iy * nx + ix; // access in rows
    unsigned int to = ix * ny + iy; // access in columns

    if (ix + 3 * blockDim.x < nx && iy < ny)
    {
        out[to]                   = in[ti];
        out[to + ny * blockDim.x]   = in[ti + blockDim.x];
        out[to + ny * 2 * blockDim.x] = in[ti + 2 * blockDim.x];
        out[to + ny * 3 * blockDim.x] = in[ti + 3 * blockDim.x];
    }
}

__global__ void transposeRow(float *out, float *in, const int nx,
                                    const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int row = iy * gridDim.x * blockDim.x + ix;

    if (row < ny)
    {
        int row_start = row * nx;
        int row_end = (row + 1) * nx;
        int col_index = row;
        for (int i = row_start; i < row_end; i++) {
            out[col_index] = in[i];
            col_index += nx;
        }
    }
}

__global__ void transposeUnroll8Row(float *out, float *in, const int nx,
                                    const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x * 8 + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    unsigned int ti = iy * nx + ix; // access in rows
    unsigned int to = ix * ny + iy; // access in columns

    if (ix + 7 * blockDim.x < nx && iy < ny)
    {
        out[to]                   = in[ti];
        out[to + ny * blockDim.x]   = in[ti + blockDim.x];
        out[to + ny * 2 * blockDim.x] = in[ti + 2 * blockDim.x];
        out[to + ny * 3 * blockDim.x] = in[ti + 3 * blockDim.x];
        out[to + ny * 4 * blockDim.x] = in[ti + 4 * blockDim.x];
        out[to + ny * 5 * blockDim.x] = in[ti + 5 * blockDim.x];
        out[to + ny * 6 * blockDim.x] = in[ti + 6 * blockDim.x];
        out[to + ny * 7 * blockDim.x] = in[ti + 7 * blockDim.x];
    }
}

// case 5 transpose kernel: read in columns and write in rows + unroll 4 blocks
__global__ void transposeUnroll4Col(float *out, float *in, const int nx,
                                    const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x * 4 + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    unsigned int ti = iy * nx + ix; // access in rows
    unsigned int to = ix * ny + iy; // access in columns

    if (ix + 3 * blockDim.x < nx && iy < ny)
    {
        out[ti]                = in[to];
        out[ti +   blockDim.x] = in[to +   blockDim.x * ny];
        out[ti + 2 * blockDim.x] = in[to + 2 * blockDim.x * ny];
        out[ti + 3 * blockDim.x] = in[to + 3 * blockDim.x * ny];
    }
}

/*
 * case 6 :  transpose kernel: read in rows and write in colunms + diagonal
 * coordinate transform
 */
__global__ void transposeDiagonalRow(float *out, float *in, const int nx,
                                     const int ny)
{
    unsigned int blk_y = blockIdx.x;
    unsigned int blk_x = (blockIdx.x + blockIdx.y) % gridDim.x;

    unsigned int ix = blockDim.x * blk_x + threadIdx.x;
    unsigned int iy = blockDim.y * blk_y + threadIdx.y;

    if (ix < nx && iy < ny)
    {
        out[ix * ny + iy] = in[iy * nx + ix];
    }
}

/*
 * case 7 :  transpose kernel: read in columns and write in row + diagonal
 * coordinate transform.
 */
__global__ void transposeDiagonalCol(float *out, float *in, const int nx,
                                     const int ny)
{
    unsigned int blk_y = blockIdx.x;
    unsigned int blk_x = (blockIdx.x + blockIdx.y) % gridDim.x;

    unsigned int ix = blockDim.x * blk_x + threadIdx.x;
    unsigned int iy = blockDim.y * blk_y + threadIdx.y;

    if (ix < nx && iy < ny)
    {
        out[iy * nx + ix] = in[ix * ny + iy];
    }
}

__global__ void transposeDiagonalColUnroll4(float *out, float *in, const int nx,
                                     const int ny)
{
    unsigned int blk_y = blockIdx.x;
    unsigned int blk_x = (blockIdx.x + blockIdx.y) % gridDim.x;

    unsigned int ix_stride = blockDim.x * blk_x;
    unsigned int ix = ix_stride * 4 + threadIdx.x;
    unsigned int iy = blockDim.y * blk_y + threadIdx.y;

    if (ix < nx && iy < ny)
    {
        out[iy * nx + ix] = in[ix * ny + iy];
        out[iy * nx + ix + blockDim.x] = in[(ix + blockDim.x) * ny + iy];
        out[iy * nx + ix + 2 * blockDim.x] =
            in[(ix + 2 * blockDim.x) * ny + iy];
        out[iy * nx + ix + 3 * blockDim.x] =
            in[(ix + 3 * blockDim.x) * ny + iy];
    }
}

// main functions
int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s starting transpose at ", argv[0]);
    printf("device %d: %s ", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up array size 2048
    int nx = 1 << 11;
    int ny = 1 << 11;

    // select a kernel and block size
    int iKernel = 0;
    int blockx = 16;
    int blocky = 16;

    if (argc > 1) iKernel = atoi(argv[1]);

    if (argc > 2) blockx  = atoi(argv[2]);

    if (argc > 3) blocky  = atoi(argv[3]);

    if (argc > 4) nx  = atoi(argv[4]);

    if (argc > 5) ny  = atoi(argv[5]);

    printf(" with matrix nx %d ny %d with kernel %d\n", nx, ny, iKernel);
    size_t nBytes = nx * ny * sizeof(float);

    // execution configuration
    dim3 block (blockx, blocky);
    dim3 grid  ((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
    dim3 grid4 ((nx + block.x - 1) / (block.x * 4), (ny + block.y - 1) /
            (block.y * 4));
    dim3 grid8 ((nx + block.x - 1) / (block.x * 8), (ny + block.y - 1) /
            (block.y * 8));

    // allocate host memory
    float *h_A = (float *)malloc(nBytes);
    float *hostRef = (float *)malloc(nBytes);
    float *gpuRef  = (float *)malloc(nBytes);

    // initialize host array
    initialData(h_A, nx * ny);

    // transpose at host side
    transposeHost(hostRef, h_A, nx, ny);

    // allocate device memory
    float *d_A, *d_C;
    CHECK(hipMalloc((float**)&d_A, nBytes));
    CHECK(hipMalloc((float**)&d_C, nBytes));

    // copy data from host to device
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));

    // warmup to avoide startup overhead
    double iStart = seconds();
    warmup<<<grid, block>>>(d_C, d_A, nx, ny);
    CHECK(hipDeviceSynchronize());
    double iElaps = seconds() - iStart;
    printf("warmup         elapsed %f sec\n", iElaps);
    CHECK(hipGetLastError());

    // transposeNaiveRow
    iStart = seconds();
    transposeNaiveRow<<<grid, block>>>(d_C, d_A, nx, ny);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;

    printf("transposeNaiveRow elapsed %f sec <<< grid (%d,%d) block (%d,%d)>>>\n", iElaps,
           grid.x, grid.y, block.x, block.y);
    CHECK(hipGetLastError());
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    checkResult(hostRef, gpuRef, nx * ny, 1);

    // transposeNaiveCol
    iStart = seconds();
    transposeNaiveCol<<<grid, block>>>(d_C, d_A, nx, ny);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;

    printf("transposeNaiveCol elapsed %f sec <<< grid (%d,%d) block (%d,%d)>>>\n", iElaps,
           grid.x, grid.y, block.x, block.y);
    CHECK(hipGetLastError());
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    checkResult(hostRef, gpuRef, nx * ny, 1);

    // transposeUnroll4Row
    iStart = seconds();
    transposeUnroll4Row<<<grid4, block>>>(d_C, d_A, nx, ny);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;

    printf("transposeUnroll4Row elapsed %f sec <<< grid (%d,%d) block (%d,%d)>>>\n", iElaps,
           grid.x, grid.y, block.x, block.y);
    CHECK(hipGetLastError());
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    checkResult(hostRef, gpuRef, nx * ny, 1);

    // transposeRow
    iStart = seconds();
    transposeRow<<<grid, block>>>(d_C, d_A, nx, ny);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;

    printf("transposeRow elapsed %f sec <<< grid (%d,%d) block (%d,%d)>>>\n", iElaps,
           grid.x, grid.y, block.x, block.y);
    CHECK(hipGetLastError());
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    checkResult(hostRef, gpuRef, nx * ny, 1);

    // transposeUnroll8Row
    iStart = seconds();
    transposeUnroll8Row<<<grid8, block>>>(d_C, d_A, nx, ny);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;

    printf("transposeUnroll8Row elapsed %f sec <<< grid (%d,%d) block (%d,%d)>>>\n", iElaps,
           grid.x, grid.y, block.x, block.y);
    CHECK(hipGetLastError());
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    checkResult(hostRef, gpuRef, nx * ny, 1);

    // transposeUnroll4Col
    iStart = seconds();
    transposeUnroll4Col<<<grid4, block>>>(d_C, d_A, nx, ny);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;

    printf("transposeUnroll4Col elapsed %f sec <<< grid (%d,%d) block (%d,%d)>>>\n", iElaps,
           grid.x, grid.y, block.x, block.y);
    CHECK(hipGetLastError());
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    checkResult(hostRef, gpuRef, nx * ny, 1);

    // transposeDiagonalRow
    iStart = seconds();
    transposeDiagonalRow<<<grid, block>>>(d_C, d_A, nx, ny);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;

    printf("transposeDiagonalRow elapsed %f sec <<< grid (%d,%d) block (%d,%d)>>>\n", iElaps,
           grid.x, grid.y, block.x, block.y);
    CHECK(hipGetLastError());
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    checkResult(hostRef, gpuRef, nx * ny, 1);

    // transposeDiagonalCol
    iStart = seconds();
    transposeDiagonalCol<<<grid, block>>>(d_C, d_A, nx, ny);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;

    printf("transposeDiagonalCol elapsed %f sec <<< grid (%d,%d) block (%d,%d)>>>\n", iElaps,
           grid.x, grid.y, block.x, block.y);
    CHECK(hipGetLastError());
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    checkResult(hostRef, gpuRef, nx * ny, 1);

    // transposeDiagonalColUnroll4
    iStart = seconds();
    transposeDiagonalColUnroll4<<<grid4, block>>>(d_C, d_A, nx, ny);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;

    printf("transposeDiagonalColUnroll4 elapsed %f sec <<< grid (%d,%d) block (%d,%d)>>>\n", iElaps,
           grid.x, grid.y, block.x, block.y);
    CHECK(hipGetLastError());
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    checkResult(hostRef, gpuRef, nx * ny, 1);


    // free host and device memory
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_C));
    free(h_A);
    free(hostRef);
    free(gpuRef);

    // reset device
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}
