#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * This example demonstrates the impact of misaligned writes on performance by
 * forcing misaligned writes to occur on a float*.
 */

void initialData(float *ip,  int size)
{
    for (int i = 0; i < size; i++)
    {
        ip[i] = (float)( rand() & 0xFF ) / 100.0f;
    }

    return;
}

__global__ void readWriteOffset(float *A, float *B, float *C, const int n,
                            int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    if (k < n) C[k] = A[k] + B[k];
}

__global__ void readWriteOffsetUnroll4(float *A, float *B, float *C,
                            const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x * 4 + threadIdx.x;
    unsigned int k = i + offset;

    if (k + 3 * blockDim.x < n)
    {
        C[k]                  = A[k]                  + B[k];
        C[k + blockDim.x]     = A[k + blockDim.x]     + B[k + blockDim.x];
        C[k + 2 * blockDim.x] = A[k + 2 * blockDim.x] + B[k + 2 * blockDim.x];
        C[k + 3 * blockDim.x] = A[k + 3 * blockDim.x] + B[k + 3 * blockDim.x];
    }
}

__global__ void warmup(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    if (k < n) C[k] = A[i] + B[i];
}

int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s starting reduction at ", argv[0]);
    printf("device %d: %s ", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up array size
    int nElem = 1 << 20; // total number of elements to reduce
    printf(" with array size %d\n", nElem);
    size_t nBytes = nElem * sizeof(float);

    // set up offset for summary
    int blocksize = 512;
    int offset = 0;

    if (argc > 1) offset    = atoi(argv[1]);

    if (argc > 2) blocksize = atoi(argv[2]);

    // execution configuration
    dim3 block (blocksize, 1);
    dim3 grid  ((nElem + block.x - 1) / block.x, 1);

    // allocate host memory
    float *h_A = (float *)malloc(nBytes);
    float *h_B = (float *)malloc(nBytes);
    float *gpuRef  = (float *)malloc(nBytes);

    // initialize host array
    initialData(h_A, nElem);
    memcpy(h_B, h_A, nBytes);

    // allocate device memory
    float *d_A, *d_B, *d_C;
    CHECK(hipMalloc((float**)&d_A, nBytes));
    CHECK(hipMalloc((float**)&d_B, nBytes));
    CHECK(hipMalloc((float**)&d_C, nBytes));

    // copy data from host to device
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_A, nBytes, hipMemcpyHostToDevice));

    // warmup
    double iStart = seconds();
    warmup<<<grid, block>>>(d_A, d_B, d_C, nElem, offset);
    CHECK(hipDeviceSynchronize());
    double iElaps = seconds() - iStart;
    printf("warmup      <<< %4d, %4d >>> offset %4d elapsed %f sec\n", grid.x,
           block.x, offset, iElaps);
    CHECK(hipGetLastError());

    // readWriteOffset
    iStart = seconds();
    readWriteOffset<<<grid, block>>>(d_A, d_B, d_C, nElem, offset);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("readWriteOffset <<< %4d, %4d >>> offset %4d elapsed %f sec\n",
           grid.x, block.x, offset, iElaps);
    CHECK(hipGetLastError());

    // readWriteOffsetUnroll4
    iStart = seconds();
    readWriteOffsetUnroll4<<<grid.x / 4, block>>>(d_A, d_B, d_C, nElem, offset);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("readWriteOffsetUnroll4 <<< %4d, %4d >>> offset %4d elapsed %f "
           "sec\n", grid.x, block.x, offset, iElaps);
    CHECK(hipGetLastError());

    // free host and device memory
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));
    free(h_A);
    free(h_B);

    // reset device
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}
