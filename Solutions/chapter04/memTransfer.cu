#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#define MEGABYTE    (1024 * 1024)

int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    CHECK(hipSetDevice(dev));

    if (argc != 2) {
        printf("usage: %s <size-in-mbs>\n", argv[0]);
        return 1;
    }

    // memory size
    int n_mbs = atoi(argv[1]);
    unsigned int nbytes = n_mbs * MEGABYTE;

    // get device information
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s starting at ", argv[0]);
    printf("device %d: %s nbyte %5.2fMB\n", dev,
           deviceProp.name, nbytes / (1024.0f * 1024.0f));

    // allocate the host memory
    double start = seconds();
    float *h_a = (float *)malloc(nbytes);
    double elapsed = seconds() - start;
    printf("Host memory allocation took %2.10f us\n", elapsed * 1000000.0);

    // allocate the device memory
    float *d_a;
    CHECK(hipMalloc((float **)&d_a, nbytes));

    // initialize the host memory
    for(unsigned int i = 0; i < nbytes / sizeof(float); i++) h_a[i] = 0.5f;

    // transfer data from the host to the device
    CHECK(hipMemcpy(d_a, h_a, nbytes, hipMemcpyHostToDevice));

    // transfer data from the device to the host
    CHECK(hipMemcpy(h_a, d_a, nbytes, hipMemcpyDeviceToHost));

    // free memory
    CHECK(hipFree(d_a));

    start = seconds();
    free(h_a);
    elapsed = seconds() - start;
    printf("Host memory deallocation took %2.10f us\n", elapsed * 1000000.0);

    // reset device
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}
