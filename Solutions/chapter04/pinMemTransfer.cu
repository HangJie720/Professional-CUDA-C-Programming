#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#define MEGABYTE    (1024 * 1024)

int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    CHECK(hipSetDevice(dev));

    if (argc != 2) {
        printf("usage: %s <size-in-mbs>\n", argv[0]);
        return 1;
    }

    // memory size
    int n_mbs = atoi(argv[1]);
    unsigned int nbytes = n_mbs * MEGABYTE;

    // get device information
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));

    if (!deviceProp.canMapHostMemory)
    {
        printf("Device %d does not support mapping CPU host memory!\n", dev);
        CHECK(hipDeviceReset());
        exit(EXIT_SUCCESS);
    }

    printf("%s starting at ", argv[0]);
    printf("device %d: %s nbyte %5.2fMB canMap %d\n", dev,
           deviceProp.name, nbytes / (1024.0f * 1024.0f),
           deviceProp.canMapHostMemory);

    // allocate pinned host memory
    float *h_a;
    double start = seconds();
    CHECK(hipHostMalloc ((float **)&h_a, nbytes));
    double elapsed = seconds() - start;
    printf("Host memory allocation took %2.10f us\n", elapsed * 1000000.0);

    // allocate device memory
    float *d_a;
    CHECK(hipMalloc((float **)&d_a, nbytes));

    // initialize host memory
    memset(h_a, 0, nbytes);

    for (int i = 0; i < nbytes / sizeof(float); i++) h_a[i] = 100.10f;

    // transfer data from the host to the device
    CHECK(hipMemcpy(d_a, h_a, nbytes, hipMemcpyHostToDevice));

    // transfer data from the device to the host
    CHECK(hipMemcpy(h_a, d_a, nbytes, hipMemcpyDeviceToHost));

    // free memory
    CHECK(hipFree(d_a));
    start = seconds();
    CHECK(hipHostFree(h_a));
    elapsed = seconds() - start;
    printf("Host memory deallocation took %2.10f us\n", elapsed * 1000000.0);

    // reset device
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}
