#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <stdio.h>
#include <stdlib.h>

/**
 * This example illustrates implementation of custom atomic operations using
 * CUDA's built-in atomicCAS function to implement atomic signed 32-bit integer
 * addition.
 **/

__device__ int myAtomicAdd(int *address, int incr)
{
    // Create an initial guess for the value stored at *address.
    int guess = *address;
    int oldValue = atomicCAS(address, guess, guess + incr);

    // Loop while the guess is incorrect.
    while (oldValue != guess)
    {
        guess = oldValue;
        oldValue = atomicCAS(address, guess, guess + incr);
    }

    return oldValue;
}

__global__ void kernel(int *sharedInteger)
{
    myAtomicAdd(sharedInteger, 1);
}

int main(int argc, char **argv)
{
    int h_sharedInteger;
    int *d_sharedInteger;
    CHECK(hipMalloc((void **)&d_sharedInteger, sizeof(int)));
    CHECK(hipMemset(d_sharedInteger, 0x00, sizeof(int)));

    kernel<<<4, 128>>>(d_sharedInteger);

    CHECK(hipMemcpy(&h_sharedInteger, d_sharedInteger, sizeof(int),
                     hipMemcpyDeviceToHost));
    printf("4 x 128 increments led to value of %d\n", h_sharedInteger);

    return 0;
}

