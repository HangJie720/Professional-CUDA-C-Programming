#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <stdio.h>
#include <stdlib.h>

/**
 * This example illustrates the effect on numerical accuracy of fusing a
 * multiply-add into a single MAD instruction.
 **/

__global__ void fmad_kernel(double x, double y, double *out)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid == 0)
    {
        *out = x * x + y;
    }
}

double host_fmad_kernel(double x, double y)
{
    return x * x + y;
}

int main(int argc, char **argv)
{
    double *d_out, h_out;
    double x = 2.891903;
    double y = -3.980364;

    double host_value = host_fmad_kernel(x, y);
    CHECK(hipMalloc((void **)&d_out, sizeof(double)));
    fmad_kernel<<<1, 32>>>(x, y, d_out);
    CHECK(hipMemcpy(&h_out, d_out, sizeof(double),
                     hipMemcpyDeviceToHost));

    if (host_value == h_out)
    {
        printf("The device output the same value as the host.\n");
    }
    else
    {
        printf("The device output a different value than the host, diff=%e.\n",
               fabs(host_value - h_out));
    }

    return 0;
}
