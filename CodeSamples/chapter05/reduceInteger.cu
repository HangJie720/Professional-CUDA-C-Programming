#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#define DIM 128

/*
 * An example of using shared memory to optimize performance of a parallel
 * reduction by constructing partial results for a thread block in shared memory
 * before flushing to global memory.
 */

extern __shared__ int dsmem[];

// Recursive Implementation of Interleaved Pair Approach
int recursiveReduce(int *data, int const size)
{
    if (size == 1) return data[0];

    int const stride = size / 2;

    for (int i = 0; i < stride; i++)
        data[i] += data[i + stride];

    return recursiveReduce(data, stride);
}

// unroll4 + complete unroll for loop + gmem
__global__ void reduceGmem(int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= n) return;

    // in-place reduction in global memory
    if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];

    __syncthreads();

    if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];

    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];

    __syncthreads();

    if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];

    __syncthreads();

    // unrolling warp
    if (tid < 32)
    {
        volatile int *vsmem = idata;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceSmem(int *g_idata, int *g_odata, unsigned int n)
{
    __shared__ int smem[DIM];

    // set thread ID
    unsigned int tid = threadIdx.x;

    // boundary check
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= n) return;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // set to smem by each threads
    smem[tid] = idata[tid];
    __syncthreads();

    // in-place reduction in shared memory
    if (blockDim.x >= 1024 && tid < 512) smem[tid] += smem[tid + 512];

    __syncthreads();

    if (blockDim.x >= 512 && tid < 256) smem[tid] += smem[tid + 256];

    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) smem[tid] += smem[tid + 128];

    __syncthreads();

    if (blockDim.x >= 128 && tid < 64)  smem[tid] += smem[tid + 64];

    __syncthreads();

    // unrolling warp
    if (tid < 32)
    {
        volatile int *vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}

__global__ void reduceSmemDyn(int *g_idata, int *g_odata, unsigned int n)
{
    extern __shared__ int smem[];

    // set thread ID
    unsigned int tid = threadIdx.x;
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // set to smem by each threads
    smem[tid] = idata[tid];
    __syncthreads();

    // in-place reduction in global memory
    if (blockDim.x >= 1024 && tid < 512)  smem[tid] += smem[tid + 512];

    __syncthreads();

    if (blockDim.x >= 512 && tid < 256)  smem[tid] += smem[tid + 256];

    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) smem[tid] += smem[tid + 128];

    __syncthreads();

    if (blockDim.x >= 128 && tid < 64) smem[tid] += smem[tid + 64];

    __syncthreads();

    // unrolling warp
    if (tid < 32)
    {
        volatile int *vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}

// unroll4 + complete unroll for loop + gmem
__global__ void reduceGmemUnroll(int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 4;

    // unrolling 4
    if (idx < n)
    {
        int a1, a2, a3, a4;
        a1 = a2 = a3 = a4 = 0;
        a1 = g_idata[idx];
        if (idx + blockDim.x < n) a2 = g_idata[idx + blockDim.x];
        if (idx + 2 * blockDim.x < n) a3 = g_idata[idx + 2 * blockDim.x];
        if (idx + 3 * blockDim.x < n) a4 = g_idata[idx + 3 * blockDim.x];
        g_idata[idx] = a1 + a2 + a3 + a4;
    }

    __syncthreads();

    // in-place reduction in global memory
    if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];

    __syncthreads();

    if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];

    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];

    __syncthreads();

    if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];

    __syncthreads();

    // unrolling warp
    if (tid < 32)
    {
        volatile int *vsmem = idata;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceSmemUnroll(int *g_idata, int *g_odata, unsigned int n)
{
    // static shared memory
    __shared__ int smem[DIM];

    // set thread ID
    unsigned int tid = threadIdx.x;

    // global index, 4 blocks of input data processed at a time
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    // unrolling 4 blocks
    int tmpSum = 0;

    // boundary check
    if (idx < n)
    {
        int a1, a2, a3, a4;
        a1 = a2 = a3 = a4 = 0;
        a1 = g_idata[idx];
        if (idx + blockDim.x < n) a2 = g_idata[idx + blockDim.x];
        if (idx + 2 * blockDim.x < n) a3 = g_idata[idx + 2 * blockDim.x];
        if (idx + 3 * blockDim.x < n) a4 = g_idata[idx + 3 * blockDim.x];
        tmpSum = a1 + a2 + a3 + a4;
    }

    smem[tid] = tmpSum;
    __syncthreads();

    // in-place reduction in shared memory
    if (blockDim.x >= 1024 && tid < 512) smem[tid] += smem[tid + 512];

    __syncthreads();

    if (blockDim.x >= 512 && tid < 256)  smem[tid] += smem[tid + 256];

    __syncthreads();

    if (blockDim.x >= 256 && tid < 128)  smem[tid] += smem[tid + 128];

    __syncthreads();

    if (blockDim.x >= 128 && tid < 64)   smem[tid] += smem[tid + 64];

    __syncthreads();

    // unrolling warp
    if (tid < 32)
    {
        volatile int *vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}

__global__ void reduceSmemUnrollDyn(int *g_idata, int *g_odata, unsigned int n)
{
    extern __shared__ int smem[];

    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    // unrolling 4
    int tmpSum = 0;

    if (idx < n)
    {
        int a1, a2, a3, a4;
        a1 = a2 = a3 = a4 = 0;
        a1 = g_idata[idx];
        if (idx + blockDim.x < n) a2 = g_idata[idx + blockDim.x];
        if (idx + 2 * blockDim.x < n) a3 = g_idata[idx + 2 * blockDim.x];
        if (idx + 3 * blockDim.x < n) a4 = g_idata[idx + 3 * blockDim.x];
        tmpSum = a1 + a2 + a3 + a4;
    }

    smem[tid] = tmpSum;
    __syncthreads();

    // in-place reduction in global memory
    if (blockDim.x >= 1024 && tid < 512)  smem[tid] += smem[tid + 512];

    __syncthreads();

    if (blockDim.x >= 512 && tid < 256)  smem[tid] += smem[tid + 256];

    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) smem[tid] += smem[tid + 128];

    __syncthreads();

    if (blockDim.x >= 128 && tid < 64) smem[tid] += smem[tid + 64];

    __syncthreads();

    // unrolling warp
    if (tid < 32)
    {
        volatile int *vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}

__global__ void reduceNeighboredGmem(int *g_idata, int *g_odata,
                                     unsigned int  n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
    if (idx >= n) return;

    // in-place reduction in global memory
    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        if ((tid % (2 * stride)) == 0)
        {
            idata[tid] += idata[tid + stride];
        }

        // synchronize within threadblock
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceNeighboredSmem(int *g_idata, int *g_odata,
                                     unsigned int  n)
{
    __shared__ int smem[DIM];

    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
    if (idx >= n) return;

    smem[tid] = idata[tid];
    __syncthreads();

    // in-place reduction in global memory
    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        if ((tid % (2 * stride)) == 0)
        {
            smem[tid] += smem[tid + stride];
        }

        // synchronize within threadblock
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}

int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s starting reduction at ", argv[0]);
    printf("device %d: %s ", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    bool bResult = false;

    // initialization
    int size = 1 << 22; // total number of elements to reduce
    printf("    with array size %d  ", size);

    // execution configuration
    int blocksize = DIM;   // initial block size

    dim3 block (blocksize, 1);
    dim3 grid  ((size + block.x - 1) / block.x, 1);
    printf("grid %d block %d\n", grid.x, block.x);

    // allocate host memory
    size_t bytes = size * sizeof(int);
    int *h_idata = (int *) malloc(bytes);
    int *h_odata = (int *) malloc(grid.x * sizeof(int));
    int *tmp     = (int *) malloc(bytes);

    // initialize the array
    for (int i = 0; i < size; i++)
    {
        h_idata[i] = (int)( rand() & 0xFF );
    }

    memcpy (tmp, h_idata, bytes);

    int gpu_sum = 0;

    // allocate device memory
    int *d_idata = NULL;
    int *d_odata = NULL;
    CHECK(hipMalloc((void **) &d_idata, bytes));
    CHECK(hipMalloc((void **) &d_odata, grid.x * sizeof(int)));

    // cpu reduction
    int cpu_sum = recursiveReduce (tmp, size);
    printf("cpu reduce          : %d\n", cpu_sum);

    // reduce gmem
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    reduceNeighboredGmem<<<grid.x, block>>>(d_idata, d_odata, size);
    CHECK(hipMemcpy(h_odata, d_odata, grid.x * sizeof(int),
                     hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];

    printf("reduceNeighboredGmem: %d <<<grid %d block %d>>>\n", gpu_sum, grid.x,
           block.x);

    // reduce gmem
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    reduceNeighboredSmem<<<grid.x, block>>>(d_idata, d_odata, size);
    CHECK(hipMemcpy(h_odata, d_odata, grid.x * sizeof(int),
                     hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];

    printf("reduceNeighboredSmem: %d <<<grid %d block %d>>>\n", gpu_sum, grid.x,
           block.x);

    // reduce gmem
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    reduceGmem<<<grid.x, block>>>(d_idata, d_odata, size);
    CHECK(hipMemcpy(h_odata, d_odata, grid.x * sizeof(int),
                     hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];

    printf("reduceGmem          : %d <<<grid %d block %d>>>\n", gpu_sum, grid.x,
           block.x);

    // reduce smem
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    reduceSmem<<<grid.x, block>>>(d_idata, d_odata, size);
    CHECK(hipMemcpy(h_odata, d_odata, grid.x * sizeof(int),
                     hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];

    printf("reduceSmem          : %d <<<grid %d block %d>>>\n", gpu_sum, grid.x,
           block.x);

    // reduce smem
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    reduceSmemDyn<<<grid.x, block, blocksize*sizeof(int)>>>(d_idata, d_odata,
            size);
    CHECK(hipMemcpy(h_odata, d_odata, grid.x * sizeof(int),
                     hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];

    printf("reduceSmemDyn       : %d <<<grid %d block %d>>>\n", gpu_sum, grid.x,
           block.x);

    // reduce gmem
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    reduceGmemUnroll<<<grid.x / 4, block>>>(d_idata, d_odata, size);
    CHECK(hipMemcpy(h_odata, d_odata, grid.x / 4 * sizeof(int),
                     hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x / 4; i++) gpu_sum += h_odata[i];

    printf("reduceGmemUnroll4   : %d <<<grid %d block %d>>>\n", gpu_sum,
            grid.x / 4, block.x);

    // reduce smem
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    reduceSmemUnroll<<<grid.x / 4, block>>>(d_idata, d_odata, size);
    CHECK(hipMemcpy(h_odata, d_odata, grid.x / 4 * sizeof(int),
                     hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x / 4; i++) gpu_sum += h_odata[i];

    printf("reduceSmemUnroll4   : %d <<<grid %d block %d>>>\n", gpu_sum,
            grid.x / 4, block.x);

    // reduce smem
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    reduceSmemUnrollDyn<<<grid.x / 4, block, DIM*sizeof(int)>>>(d_idata,
            d_odata, size);
    CHECK(hipMemcpy(h_odata, d_odata, grid.x / 4 * sizeof(int),
                     hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x / 4; i++) gpu_sum += h_odata[i];

    printf("reduceSmemDynUnroll4: %d <<<grid %d block %d>>>\n", gpu_sum,
            grid.x / 4, block.x);

    // free host memory
    free(h_idata);
    free(h_odata);

    // free device memory
    CHECK(hipFree(d_idata));
    CHECK(hipFree(d_odata));

    // reset device
    CHECK(hipDeviceReset());

    // check the results
    bResult = (gpu_sum == cpu_sum);

    if(!bResult) printf("Test failed!\n");

    return EXIT_SUCCESS;
}
