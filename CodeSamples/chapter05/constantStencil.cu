#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * An example of using constant memory to optimize performance of a stencil
 * computation by storing coefficients of the computation in a constant memory
 * array (coef).
 */

#define RADIUS 4
#define BDIM 32

// constant memory
__constant__ float coef[RADIUS + 1];

// FD coeffecient
#define a0     0.00000f
#define a1     0.80000f
#define a2    -0.20000f
#define a3     0.03809f
#define a4    -0.00357f

void initialData(float *in,  const int size)
{
    for (int i = 0; i < size; i++)
    {
        in[i] = (float)(rand() & 0xFF) / 100.0f;
    }
}

void printData(float *in,  const int size)
{
    for (int i = RADIUS; i < size; i++)
    {
        printf("%f ", in[i]);
    }

    printf("\n");
}

void setup_coef_constant (void)
{
    const float h_coef[] = {a0, a1, a2, a3, a4};
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL( coef), h_coef, (RADIUS + 1) * sizeof(float)));
}

void cpu_stencil_1d (float *in, float *out, int isize)
{
    for (int i = RADIUS; i <= isize; i++)
    {
        float tmp = a1 * (in[i + 1] - in[i - 1])
                    + a2 * (in[i + 2] - in[i - 2])
                    + a3 * (in[i + 3] - in[i - 3])
                    + a4 * (in[i + 4] - in[i - 4]);
        out[i] = tmp;
    }
}

void checkResult(float *hostRef, float *gpuRef, const int size)
{
    double epsilon = 1.0E-6;
    bool match = 1;

    for (int i = RADIUS; i < size; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("different on %dth element: host %f gpu %f\n", i, hostRef[i],
                   gpuRef[i]);
            break;
        }
    }

    if (!match) printf("Arrays do not match.\n\n");
}

__global__ void stencil_1d(float *in, float *out, int N)
{
    // shared memory
    __shared__ float smem[BDIM + 2 * RADIUS];

    // index to global memory
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    while (idx < N)
    {

        // index to shared memory for stencil calculatioin
        int sidx = threadIdx.x + RADIUS;

        // Read data from global memory into shared memory
        smem[sidx] = in[idx];

        // read halo part to shared memory
        if (threadIdx.x < RADIUS)
        {
            smem[sidx - RADIUS] = in[idx - RADIUS];
            smem[sidx + BDIM] = in[idx + BDIM];
        }

        // Synchronize (ensure all the data is available)
        __syncthreads();

        // Apply the stencil
        float tmp = 0.0f;

#pragma unroll
        for (int i = 1; i <= RADIUS; i++)
        {
            tmp += coef[i] * (smem[sidx + i] - smem[sidx - i]);
        }

        // Store the result
        out[idx] = tmp;

        idx += gridDim.x * blockDim.x;
    }
}


int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s starting transpose at ", argv[0]);
    printf("device %d: %s ", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up data size
    int isize = 1 << 24;

    size_t nBytes = (isize + 2 * RADIUS) * sizeof(float);
    printf("array size: %d ", isize);

    bool iprint = 0;

    // allocate host memory
    float *h_in    = (float *)malloc(nBytes);
    float *hostRef = (float *)malloc(nBytes);
    float *gpuRef  = (float *)malloc(nBytes);

    // allocate device memory
    float *d_in, *d_out;
    CHECK(hipMalloc((float**)&d_in, nBytes));
    CHECK(hipMalloc((float**)&d_out, nBytes));

    // initialize host array
    initialData(h_in, isize + 2 * RADIUS);

    // Copy to device
    CHECK(hipMemcpy(d_in, h_in, nBytes, hipMemcpyHostToDevice));

    // set up constant memory
    setup_coef_constant();

    // launch configuration
    hipDeviceProp_t info;
    CHECK(hipGetDeviceProperties(&info, 0));
    dim3 block(BDIM, 1);
    dim3 grid(info.maxGridSize[0] < isize / block.x ? info.maxGridSize[0] :
            isize / block.x, 1);
    printf("(grid, block) %d,%d \n ", grid.x, block.x);

    // Launch stencil_1d() kernel on GPU
    stencil_1d<<<grid, block>>>(d_in + RADIUS, d_out + RADIUS, isize);

    // Copy result back to host
    CHECK(hipMemcpy(gpuRef, d_out, nBytes, hipMemcpyDeviceToHost));

    // apply cpu stencil
    cpu_stencil_1d(h_in, hostRef, isize);

    // check results
    checkResult(hostRef, gpuRef, isize);

    // print out results
    if(iprint)
    {
        printData(gpuRef, isize);
        printData(hostRef, isize);
    }

    // Cleanup
    CHECK(hipFree(d_in));
    CHECK(hipFree(d_out));
    free(h_in);
    free(hostRef);
    free(gpuRef);

    // reset device
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}
