#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2012, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived 
 *    from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * Paulius Micikevicius (pauliusm@nvidia.com)
 * Max Grossman (jmaxg3@gmail.com)
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <unistd.h>
#include "common.h"
#include "common2d.h"

#define BDIMX   32
#define BDIMY   16
#define SHAREDX(radius) (BDIMX + 2 * (radius))
#define SHAREDY(radius) (BDIMY + 2 * (radius))
#define CACHE_INDEX(y, x, radius)   ((y) * SHAREDX(radius) + (x))

#ifndef PERC_CPU
#define PERC_CPU    0.5
#endif

__constant__ TYPE const_c_coeff[NUM_COEFF];

__global__ void fwd_kernel(TYPE *next, TYPE *curr, TYPE *vsq,
        int nx, int ny, int dimx, int radius) {
    extern __shared__ TYPE cache[];

    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int this_offset = POINT_OFFSET(x, y, dimx, radius);
    const int this_y = radius + threadIdx.y;
    const int this_x = radius + threadIdx.x;

    cache[CACHE_INDEX(this_y, this_x, radius)] =
        curr[POINT_OFFSET(x, y, dimx, radius)];
    if (threadIdx.y < radius) {
        cache[CACHE_INDEX(threadIdx.y, this_x, radius)] =
            curr[POINT_OFFSET(x, y - radius, dimx, radius)];
    }
    if (threadIdx.y >= radius && threadIdx.y < 2 * radius) {
        cache[CACHE_INDEX(threadIdx.y + blockDim.y, this_x, radius)] =
            curr[POINT_OFFSET(x, y - radius + blockDim.y, dimx, radius)];
    }
    if (threadIdx.x < radius) {
        cache[CACHE_INDEX(this_y, threadIdx.x, radius)] =
            curr[POINT_OFFSET(x - radius, y, dimx, radius)];
    }
    if (threadIdx.x >= radius && threadIdx.x < 2 * radius) {
        cache[CACHE_INDEX(this_y, threadIdx.x + blockDim.x, radius)] =
            curr[POINT_OFFSET(x - radius + blockDim.x, y, dimx, radius)];
    }

    __syncthreads();

    TYPE temp = 2.0f * cache[CACHE_INDEX(this_y, this_x, radius)] - next[this_offset];
    TYPE div = const_c_coeff[0] * cache[CACHE_INDEX(this_y, this_x, radius)];

    for (int d = radius; d >= 1; d--) {
        div += const_c_coeff[d] * (cache[CACHE_INDEX(this_y + d, this_x, radius)] +
                cache[CACHE_INDEX(this_y - d, this_x, radius)] + cache[CACHE_INDEX(this_y, this_x + d, radius)] +
                cache[CACHE_INDEX(this_y, this_x - d, radius)]);
    }
    next[this_offset] = temp + div * vsq[this_offset];
}

static void fwd(TYPE *next, TYPE *curr, TYPE *vsq,
        TYPE *c_coeff, int nx, int ny, int dimx, int radius) {

#pragma omp parallel for collapse(2)
    for (int y = 0; y < ny; y++) {
        for (int x = 0; x < nx; x++) {
            int this_offset = POINT_OFFSET(x, y, dimx, radius);
            TYPE temp = 2.0f * curr[this_offset] - next[this_offset];
            TYPE div = c_coeff[0] * curr[this_offset];
            for (int d = 1; d <= radius; d++) {
                int y_pos_offset = POINT_OFFSET(x, y + d, dimx, radius);
                int y_neg_offset = POINT_OFFSET(x, y - d, dimx, radius);
                int x_pos_offset = POINT_OFFSET(x + d, y, dimx, radius);
                int x_neg_offset = POINT_OFFSET(x - d, y, dimx, radius);
                div += c_coeff[d] * (curr[y_pos_offset] +
                        curr[y_neg_offset] + curr[x_pos_offset] +
                        curr[x_neg_offset]);
            }
            next[this_offset] = temp + div * vsq[this_offset];
        }
    }
}

int main(int argc, char *argv[]) {
    config conf;
    setup_config(&conf, argc, argv);
    init_progress(conf.progress_width, conf.nsteps, conf.progress_disabled);

#ifndef PADDING
    fprintf(stderr, "Must be compiled with -DPADDING\n");
    return 1;
#endif

    if (conf.nx % BDIMX != 0) {
        fprintf(stderr, "Invalid nx configuration, must be an even multiple of "
                "%d\n", BDIMX);
        return 1;
    }
    if (conf.ny % BDIMY != 0) {
        fprintf(stderr, "Invalid ny configuration, must be an even multiple of "
                "%d\n", BDIMY);
        return 1;
    }
    if (conf.radius > TRANSACTION_LEN) {
        fprintf(stderr, "Radius must be less than TRANSACTION_LEN to include "
                "it in dimx padding\n");
        return 1;
    }

    TYPE dx = 20.f;
    TYPE dt = 0.002f;

    // compute the pitch for perfect coalescing
    size_t dimx = TRANSACTION_LEN + conf.nx + conf.radius;
    dimx += (TRANSACTION_LEN - (dimx % TRANSACTION_LEN));
    size_t dimy = conf.ny + 2*conf.radius;
    size_t nbytes = dimx * dimy * sizeof(TYPE);

    if (conf.verbose) {
        printf("x = %zu, y = %zu\n", dimx, dimy);
        printf("nsteps = %d\n", conf.nsteps);
        printf("radius = %d\n", conf.radius);
    }

    TYPE c_coeff[NUM_COEFF];
    TYPE *curr, *next, *vsq;
    CHECK(hipHostMalloc((void **)&curr, nbytes));
    CHECK(hipHostMalloc((void **)&next, nbytes));
    CHECK(hipHostMalloc((void **)&vsq, nbytes));

    config_sources(&conf.srcs, &conf.nsrcs, conf.nx, conf.ny, conf.nsteps);
    TYPE **srcs = sample_sources(conf.srcs, conf.nsrcs, conf.nsteps, dt);

    init_data(curr, next, vsq, c_coeff, dimx, dimy, dx, dt);

    int ny_on_cpu = PERC_CPU * conf.ny;
    int ny_on_gpu = conf.ny - ny_on_cpu;
    printf("Processing %d layers on the GPU, %d on the CPU\n", ny_on_gpu,
            ny_on_cpu);

    TYPE *d_curr, *d_next, *d_vsq;
    CHECK(hipMalloc((void **)&d_curr, nbytes));
    CHECK(hipMalloc((void **)&d_next, nbytes));
    CHECK(hipMalloc((void **)&d_vsq, nbytes));

    dim3 block(BDIMX, BDIMY);
    dim3 grid(conf.nx / block.x, conf.ny / block.y);

    double mem_start = seconds();

    CHECK(hipMemcpy(d_curr, curr, nbytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_next, next, nbytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_vsq, vsq, nbytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(const_c_coeff), c_coeff, NUM_COEFF * sizeof(TYPE)));
    double start = seconds();
    for (int step = 0; step < conf.nsteps; step++) {
        for (int src = 0; src < conf.nsrcs; src++) {
            if (conf.srcs[src].t > step) continue;
            int src_offset = POINT_OFFSET(conf.srcs[src].x, conf.srcs[src].y,
                    dimx, conf.radius);
            CHECK(hipMemcpy(d_curr + src_offset, srcs[src] + step,
                        sizeof(TYPE), hipMemcpyHostToDevice));
            curr[src_offset] = srcs[src][step];
        }

        fwd_kernel<<<grid, block, SHAREDY(conf.radius) * SHAREDX(conf.radius) *
            sizeof(TYPE)>>>(d_next, d_curr, d_vsq, conf.nx, conf.ny, dimx, conf.radius);
        fwd(next + (ny_on_gpu * dimx), curr + (ny_on_gpu * dimx),
                vsq + (ny_on_gpu * dimx), c_coeff, conf.nx, ny_on_cpu, dimx, conf.radius);

        CHECK(hipMemcpy(next + (ny_on_gpu * dimx), d_next + (ny_on_gpu * dimx),
                    conf.radius * dimx * sizeof(TYPE), hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(d_next + ((ny_on_gpu + conf.radius) * dimx),
                    next + ((ny_on_gpu + conf.radius) * dimx),
                    conf.radius * dimx * sizeof(TYPE), hipMemcpyHostToDevice));

        TYPE *tmp = d_next;
        d_next = d_curr;
        d_curr = tmp;

        tmp = next;
        next = curr;
        curr = tmp;

        update_progress(step + 1);
    }
    CHECK(hipDeviceSynchronize());
    double compute_s = seconds() - start;

    CHECK(hipMemcpy(curr, d_curr, (conf.radius + ny_on_gpu) * dimx * sizeof(TYPE),
                hipMemcpyDeviceToHost));
    double total_s = seconds() - mem_start;

    finish_progress();

    float point_rate = (float)conf.nx * conf.ny / (compute_s / conf.nsteps);
    fprintf(stderr, "iso_r4_2x:   %8.10f s total, %8.10f s/step, %8.2f Mcells/s/step\n",
            total_s, compute_s / conf.nsteps, point_rate / 1000000.f);

    if (conf.save_text) {
        save_text(curr, dimx, dimy, conf.ny, conf.nx,
                 "snap.text", conf.radius);
    }

    CHECK(hipHostFree(curr));
    CHECK(hipHostFree(next));
    CHECK(hipHostFree(vsq));
    for (int i = 0; i < conf.nsrcs; i++) {
        free(srcs[i]);
    }
    free(srcs);

    CHECK(hipFree(d_curr));
    CHECK(hipFree(d_next));
    CHECK(hipFree(d_vsq));

    return 0;
}
