#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * This example demonstrates the impact of misaligned writes on performance by
 * forcing misaligned writes to occur on a float*.
 */

void checkResult(float *hostRef, float *gpuRef, const int N, const int offset)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = offset; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("different on %dth element: host %f gpu %f\n", i, hostRef[i],
                    gpuRef[i]);
            break;
        }
    }

    if (!match)  printf("Arrays do not match.\n\n");
}

void initialData(float *ip,  int size)
{
    for (int i = 0; i < size; i++)
    {
        ip[i] = (float)( rand() & 0xFF ) / 100.0f;
    }

    return;
}

void sumArraysOnHost(float *A, float *B, float *C, const int n, int offset)
{
    for (int idx = offset, k = 0; idx < n; idx++, k++)
    {
        C[idx] = A[k] + B[k];
    }
}

__global__ void writeOffset(float *A, float *B, float *C, const int n,
                            int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    if (k < n) C[k] = A[i] + B[i];
}

__global__ void warmup(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    if (k < n) C[k] = A[i] + B[i];
}

__global__ void writeOffsetUnroll2(float *A, float *B, float *C, const int n,
                                   int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    if (k + blockDim.x < n)
    {
        C[k]            = A[i]            + B[i];
        C[k + blockDim.x] = A[i + blockDim.x] + B[i + blockDim.x];
    }
}

__global__ void writeOffsetUnroll4(float *A, float *B, float *C, const int n,
                                   int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    if (k + 3 * blockDim.x < n)
    {
        C[k]              = A[i]              + B[i];
        C[k + blockDim.x]   = A[i +  blockDim.x] + B[i +  blockDim.x];
        C[k + 2 * blockDim.x] = A[i + 2 * blockDim.x] + B[i + 2 * blockDim.x];
        C[k + 3 * blockDim.x] = A[i + 3 * blockDim.x] + B[i + 3 * blockDim.x];
    }
}

int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s starting reduction at ", argv[0]);
    printf("device %d: %s ", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up array size
    int nElem = 1 << 20; // total number of elements to reduce
    printf(" with array size %d\n", nElem);
    size_t nBytes = nElem * sizeof(float);

    // set up offset for summary
    int blocksize = 512;
    int offset = 0;

    if (argc > 1) offset    = atoi(argv[1]);

    if (argc > 2) blocksize = atoi(argv[2]);

    // execution configuration
    dim3 block (blocksize, 1);
    dim3 grid  ((nElem + block.x - 1) / block.x, 1);

    // allocate host memory
    float *h_A = (float *)malloc(nBytes);
    float *h_B = (float *)malloc(nBytes);
    float *hostRef = (float *)malloc(nBytes);
    float *gpuRef  = (float *)malloc(nBytes);

    // initialize host array
    initialData(h_A, nElem);
    memcpy(h_B, h_A, nBytes);

    // summary at host side
    sumArraysOnHost(h_A, h_B, hostRef, nElem, offset);

    // allocate device memory
    float *d_A, *d_B, *d_C;
    CHECK(hipMalloc((float**)&d_A, nBytes));
    CHECK(hipMalloc((float**)&d_B, nBytes));
    CHECK(hipMalloc((float**)&d_C, nBytes));

    // copy data from host to device
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_A, nBytes, hipMemcpyHostToDevice));

    // warmup
    double iStart = seconds();
    warmup<<<grid, block>>>(d_A, d_B, d_C, nElem, offset);
    CHECK(hipDeviceSynchronize());
    double iElaps = seconds() - iStart;
    printf("warmup      <<< %4d, %4d >>> offset %4d elapsed %f sec\n", grid.x,
           block.x, offset, iElaps);
    CHECK(hipGetLastError());

    // kernel 1:
    iStart = seconds();
    writeOffset<<<grid, block>>>(d_A, d_B, d_C, nElem, offset);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("writeOffset <<< %4d, %4d >>> offset %4d elapsed %f sec\n", grid.x,
           block.x, offset, iElaps);
    CHECK(hipGetLastError());

    // copy kernel result back to host side and check device results
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    checkResult(hostRef, gpuRef, nElem, offset);

    // kernel 2
    iStart = seconds();
    writeOffsetUnroll2<<<grid.x / 2, block>>>(d_A, d_B, d_C, nElem / 2, offset);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("unroll2     <<< %4d, %4d >>> offset %4d elapsed %f sec\n",
            grid.x / 2, block.x, offset, iElaps);
    CHECK(hipGetLastError());

    // copy kernel result back to host side and check device results
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    checkResult(hostRef, gpuRef, nElem, offset);

    // kernel 2
    iStart = seconds();
    writeOffsetUnroll4<<<grid.x / 4, block>>>(d_A, d_B, d_C, nElem / 2, offset);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("unroll4     <<< %4d, %4d >>> offset %4d elapsed %f sec\n",
            grid.x / 4, block.x, offset, iElaps);
    CHECK(hipGetLastError());

    // copy kernel result back to host side and check device results
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    checkResult(hostRef, gpuRef, nElem, offset);

    // free host and device memory
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));
    free(h_A);
    free(h_B);

    // reset device
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}
