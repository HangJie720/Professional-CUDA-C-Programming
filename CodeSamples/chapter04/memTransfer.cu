#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * An example of using CUDA's memory copy API to transfer data to and from the
 * device. In this case, hipMalloc is used to allocate memory on the GPU and
 * hipMemcpy is used to transfer the contents of host memory to an array
 * allocated using hipMalloc.
 */

int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    CHECK(hipSetDevice(dev));

    // memory size
    unsigned int isize = 1 << 22;
    unsigned int nbytes = isize * sizeof(float);

    // get device information
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s starting at ", argv[0]);
    printf("device %d: %s memory size %d nbyte %5.2fMB\n", dev,
           deviceProp.name, isize, nbytes / (1024.0f * 1024.0f));

    // allocate the host memory
    float *h_a = (float *)malloc(nbytes);

    // allocate the device memory
    float *d_a;
    CHECK(hipMalloc((float **)&d_a, nbytes));

    // initialize the host memory
    for(unsigned int i = 0; i < isize; i++) h_a[i] = 0.5f;

    // transfer data from the host to the device
    CHECK(hipMemcpy(d_a, h_a, nbytes, hipMemcpyHostToDevice));

    // transfer data from the device to the host
    CHECK(hipMemcpy(h_a, d_a, nbytes, hipMemcpyDeviceToHost));

    // free memory
    CHECK(hipFree(d_a));
    free(h_a);

    // reset device
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}
