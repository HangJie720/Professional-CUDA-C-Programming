#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * This example demonstrates the impact of misaligned reads on performance by
 * forcing misaligned reads to occur on a float*. Kernels that reduce the
 * performance impact of misaligned reads via unrolling are also included below.
 */

void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("different on %dth element: host %f gpu %f\n", i, hostRef[i],
                    gpuRef[i]);
            break;
        }
    }

    if (!match)  printf("Arrays do not match.\n\n");
}

void initialData(float *ip,  int size)
{
    for (int i = 0; i < size; i++)
    {
        ip[i] = (float)( rand() & 0xFF ) / 100.0f;
    }

    return;
}


void sumArraysOnHost(float *A, float *B, float *C, const int n, int offset)
{
    for (int idx = offset, k = 0; idx < n; idx++, k++)
    {
        C[k] = A[idx] + B[idx];
    }
}

__global__ void warmup(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    if (k < n) C[i] = A[k] + B[k];
}

__global__ void readOffset(float *A, float *B, float *C, const int n,
                           int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    if (k < n) C[i] = A[k] + B[k];
}

__global__ void readOffsetUnroll2(float *A, float *B, float *C, const int n,
                                  int offset)
{
    unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    unsigned int k = i + offset;

    if (k < n) C[i] = A[k] + B[k];
    if (k + blockDim.x < n) {
        C[i + blockDim.x] = A[k + blockDim.x] + B[k + blockDim.x];
    }
}

__global__ void readOffsetUnroll4(float *A, float *B, float *C, const int n,
                                  int offset)
{
    unsigned int i = blockIdx.x * blockDim.x * 4 + threadIdx.x;
    unsigned int k = i + offset;

    if (k < n) C[i] = A[k]                  + B[k];
    if (k + blockDim.x < n) {
        C[i + blockDim.x]     = A[k + blockDim.x]     + B[k + blockDim.x];
    }
    if (k + 2 * blockDim.x < n) {
        C[i + 2 * blockDim.x] = A[k + 2 * blockDim.x] + B[k + 2 * blockDim.x];
    }
    if (k + 3 * blockDim.x < n) {
        C[i + 3 * blockDim.x] = A[k + 3 * blockDim.x] + B[k + 3 * blockDim.x];
    }
}

int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s starting reduction at ", argv[0]);
    printf("device %d: %s ", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up array size
    int power = 20;
    int blocksize = 512;
    int offset = 0;

    if (argc > 1) offset       = atoi(argv[1]);
    if (argc > 2) blocksize    = atoi(argv[2]);
    if (argc > 3) power        = atoi(argv[3]);

    int nElem = 1 << power; // total number of elements to reduce
    printf(" with array size %d\n", nElem);
    size_t nBytes = nElem * sizeof(float);

    // execution configuration
    dim3 block (blocksize, 1);
    dim3 grid  ((nElem + block.x - 1) / block.x, 1);

    // allocate host memory
    float *h_A = (float *)malloc(nBytes);
    float *h_B = (float *)malloc(nBytes);
    float *hostRef = (float *)malloc(nBytes);
    float *gpuRef  = (float *)malloc(nBytes);

    //  initialize host array
    initialData(h_A, nElem);
    memcpy(h_B, h_A, nBytes);

    //  summary at host side
    sumArraysOnHost(h_A, h_B, hostRef, nElem, offset);

    // allocate device memory
    float *d_A, *d_B, *d_C;
    CHECK(hipMalloc((float**)&d_A, nBytes));
    CHECK(hipMalloc((float**)&d_B, nBytes));
    CHECK(hipMalloc((float**)&d_C, nBytes));

    // copy data from host to device
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_A, nBytes, hipMemcpyHostToDevice));

    //  kernel 1:
    double iStart = seconds();
    warmup<<<grid, block>>>(d_A, d_B, d_C, nElem, offset);
    CHECK(hipDeviceSynchronize());
    double iElaps = seconds() - iStart;
    printf("warmup     <<< %4d, %4d >>> offset %4d elapsed %f sec\n", grid.x,
           block.x, offset, iElaps);
    CHECK(hipGetLastError());
    CHECK(hipMemset(d_C, 0x00, nBytes));

    // kernel 1
    iStart = seconds();
    readOffset<<<grid, block>>>(d_A, d_B, d_C, nElem, offset);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("readOffset <<< %4d, %4d >>> offset %4d elapsed %f sec\n", grid.x,
            block.x, offset, iElaps);
    CHECK(hipGetLastError());

    // copy kernel result back to host side and check device results
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    checkResult(hostRef, gpuRef, nElem-offset);
    CHECK(hipMemset(d_C, 0x00, nBytes));

    // kernel 2
    iStart = seconds();
    readOffsetUnroll2<<<grid.x/2, block>>>(d_A, d_B, d_C, nElem, offset);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("unroll2    <<< %4d, %4d >>> offset %4d elapsed %f sec\n",
            grid.x / 2, block.x, offset, iElaps);
    CHECK(hipGetLastError());

    // copy kernel result back to host side and check device results
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    checkResult(hostRef, gpuRef, nElem - offset);
    CHECK(hipMemset(d_C, 0x00, nBytes));

    // kernel 3
    iStart = seconds();
    readOffsetUnroll4<<<grid.x / 4, block>>>(d_A, d_B, d_C, nElem, offset);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("unroll4    <<< %4d, %4d >>> offset %4d elapsed %f sec\n",
            grid.x / 4, block.x, offset, iElaps);
    CHECK(hipGetLastError());

    // copy kernel result back to host side and check device results
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    checkResult(hostRef, gpuRef, nElem - offset);
    CHECK(hipMemset(d_C, 0x00, nBytes));

    // free host and device memory
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));
    free(h_A);
    free(h_B);

    // reset device
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}
