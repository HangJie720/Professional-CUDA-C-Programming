#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Various memory access pattern optimizations applied to a matrix transpose
 * kernel.
 */

#define BDIMX 16
#define BDIMY 16

void initialData(float *in,  const int size)
{
    for (int i = 0; i < size; i++)
    {
        in[i] = (float)( rand() & 0xFF ) / 10.0f; //100.0f;
    }

    return;
}

void printData(float *in,  const int size)
{
    for (int i = 0; i < size; i++)
    {
        printf("%dth element: %f\n", i, in[i]);
    }

    return;
}

void checkResult(float *hostRef, float *gpuRef, const int size, int showme)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < size; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("different on %dth element: host %f gpu %f\n", i, hostRef[i],
                    gpuRef[i]);
            break;
        }

        if (showme && i > size / 2 && i < size / 2 + 5)
        {
            // printf("%dth element: host %f gpu %f\n",i,hostRef[i],gpuRef[i]);
        }
    }

    if (!match)  printf("Arrays do not match.\n\n");
}

void transposeHost(float *out, float *in, const int nx, const int ny)
{
    for( int iy = 0; iy < ny; ++iy)
    {
        for( int ix = 0; ix < nx; ++ix)
        {
            out[ix * ny + iy] = in[iy * nx + ix];
        }
    }
}

__global__ void warmup(float *out, float *in, const int nx, const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix < nx && iy < ny)
    {
        out[iy * nx + ix] = in[iy * nx + ix];
    }
}

// case 0 copy kernel: access data in rows
__global__ void copyRow(float *out, float *in, const int nx, const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix < nx && iy < ny)
    {
        out[iy * nx + ix] = in[iy * nx + ix];
    }
}

// case 1 copy kernel: access data in columns
__global__ void copyCol(float *out, float *in, const int nx, const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix < nx && iy < ny)
    {
        out[ix * ny + iy] = in[ix * ny + iy];
    }
}

// case 2 transpose kernel: read in rows and write in columns
__global__ void transposeNaiveRow(float *out, float *in, const int nx,
                                  const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix < nx && iy < ny)
    {
        out[ix * ny + iy] = in[iy * nx + ix];
    }
}

// case 3 transpose kernel: read in columns and write in rows
__global__ void transposeNaiveCol(float *out, float *in, const int nx,
                                  const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix < nx && iy < ny)
    {
        out[iy * nx + ix] = in[ix * ny + iy];
    }
}

// case 4 transpose kernel: read in rows and write in columns + unroll 4 blocks
__global__ void transposeUnroll4Row(float *out, float *in, const int nx,
                                    const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x * 4 + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    unsigned int ti = iy * nx + ix; // access in rows
    unsigned int to = ix * ny + iy; // access in columns

    if (ix + 3 * blockDim.x < nx && iy < ny)
    {
        out[to]                   = in[ti];
        out[to + ny * blockDim.x]   = in[ti + blockDim.x];
        out[to + ny * 2 * blockDim.x] = in[ti + 2 * blockDim.x];
        out[to + ny * 3 * blockDim.x] = in[ti + 3 * blockDim.x];
    }
}

// case 5 transpose kernel: read in columns and write in rows + unroll 4 blocks
__global__ void transposeUnroll4Col(float *out, float *in, const int nx,
                                    const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x * 4 + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    unsigned int ti = iy * nx + ix; // access in rows
    unsigned int to = ix * ny + iy; // access in columns

    if (ix + 3 * blockDim.x < nx && iy < ny)
    {
        out[ti]                = in[to];
        out[ti +   blockDim.x] = in[to +   blockDim.x * ny];
        out[ti + 2 * blockDim.x] = in[to + 2 * blockDim.x * ny];
        out[ti + 3 * blockDim.x] = in[to + 3 * blockDim.x * ny];
    }
}

/*
 * case 6 :  transpose kernel: read in rows and write in colunms + diagonal
 * coordinate transform
 */
__global__ void transposeDiagonalRow(float *out, float *in, const int nx,
                                     const int ny)
{
    unsigned int blk_y = blockIdx.x;
    unsigned int blk_x = (blockIdx.x + blockIdx.y) % gridDim.x;

    unsigned int ix = blockDim.x * blk_x + threadIdx.x;
    unsigned int iy = blockDim.y * blk_y + threadIdx.y;

    if (ix < nx && iy < ny)
    {
        out[ix * ny + iy] = in[iy * nx + ix];
    }
}

/*
 * case 7 :  transpose kernel: read in columns and write in row + diagonal
 * coordinate transform.
 */
__global__ void transposeDiagonalCol(float *out, float *in, const int nx,
                                     const int ny)
{
    unsigned int blk_y = blockIdx.x;
    unsigned int blk_x = (blockIdx.x + blockIdx.y) % gridDim.x;

    unsigned int ix = blockDim.x * blk_x + threadIdx.x;
    unsigned int iy = blockDim.y * blk_y + threadIdx.y;

    if (ix < nx && iy < ny)
    {
        out[iy * nx + ix] = in[ix * ny + iy];
    }
}

// main functions
int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s starting transpose at ", argv[0]);
    printf("device %d: %s ", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up array size 2048
    int nx = 1 << 11;
    int ny = 1 << 11;

    // select a kernel and block size
    int iKernel = 0;
    int blockx = 16;
    int blocky = 16;

    if (argc > 1) iKernel = atoi(argv[1]);

    if (argc > 2) blockx  = atoi(argv[2]);

    if (argc > 3) blocky  = atoi(argv[3]);

    if (argc > 4) nx  = atoi(argv[4]);

    if (argc > 5) ny  = atoi(argv[5]);

    printf(" with matrix nx %d ny %d with kernel %d\n", nx, ny, iKernel);
    size_t nBytes = nx * ny * sizeof(float);

    // execution configuration
    dim3 block (blockx, blocky);
    dim3 grid  ((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    // allocate host memory
    float *h_A = (float *)malloc(nBytes);
    float *hostRef = (float *)malloc(nBytes);
    float *gpuRef  = (float *)malloc(nBytes);

    // initialize host array
    initialData(h_A, nx * ny);

    // transpose at host side
    transposeHost(hostRef, h_A, nx, ny);

    // allocate device memory
    float *d_A, *d_C;
    CHECK(hipMalloc((float**)&d_A, nBytes));
    CHECK(hipMalloc((float**)&d_C, nBytes));

    // copy data from host to device
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));

    // warmup to avoide startup overhead
    double iStart = seconds();
    warmup<<<grid, block>>>(d_C, d_A, nx, ny);
    CHECK(hipDeviceSynchronize());
    double iElaps = seconds() - iStart;
    printf("warmup         elapsed %f sec\n", iElaps);
    CHECK(hipGetLastError());

    // kernel pointer and descriptor
    void (*kernel)(float *, float *, int, int);
    char *kernelName;

    // set up kernel
    switch (iKernel)
    {
    case 0:
        kernel = &copyRow;
        kernelName = "CopyRow       ";
        break;

    case 1:
        kernel = &copyCol;
        kernelName = "CopyCol       ";
        break;

    case 2:
        kernel = &transposeNaiveRow;
        kernelName = "NaiveRow      ";
        break;

    case 3:
        kernel = &transposeNaiveCol;
        kernelName = "NaiveCol      ";
        break;

    case 4:
        kernel = &transposeUnroll4Row;
        kernelName = "Unroll4Row    ";
        grid.x = (nx + block.x * 4 - 1) / (block.x * 4);
        break;

    case 5:
        kernel = &transposeUnroll4Col;
        kernelName = "Unroll4Col    ";
        grid.x = (nx + block.x * 4 - 1) / (block.x * 4);
        break;

    case 6:
        kernel = &transposeDiagonalRow;
        kernelName = "DiagonalRow   ";
        break;

    case 7:
        kernel = &transposeDiagonalCol;
        kernelName = "DiagonalCol   ";
        break;
    }

    // run kernel
    iStart = seconds();
    kernel<<<grid, block>>>(d_C, d_A, nx, ny);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;

    // calculate effective_bandwidth
    float ibnd = 2 * nx * ny * sizeof(float) / 1e9 / iElaps;
    printf("%s elapsed %f sec <<< grid (%d,%d) block (%d,%d)>>> effective "
           "bandwidth %f GB\n", kernelName, iElaps, grid.x, grid.y, block.x,
           block.y, ibnd);
    CHECK(hipGetLastError());

    // check kernel results
    if (iKernel > 1)
    {
        CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
        checkResult(hostRef, gpuRef, nx * ny, 1);
    }

    // free host and device memory
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_C));
    free(h_A);
    free(hostRef);
    free(gpuRef);

    // reset device
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}
