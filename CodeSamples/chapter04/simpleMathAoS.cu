#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * A simple example of using an array of structures to store data on the device.
 * This example is used to study the impact on performance of data layout on the
 * GPU.
 *
 * AoS: one contiguous 64-bit read to get x and y (up to 300 cycles)
 */

#define LEN 1<<22

struct innerStruct
{
    float x;
    float y;
};

struct innerArray
{
    float x[LEN];
    float y[LEN];
};

void initialInnerStruct(innerStruct *ip,  int size)
{
    for (int i = 0; i < size; i++)
    {
        ip[i].x = (float)(rand() & 0xFF) / 100.0f;
        ip[i].y = (float)(rand() & 0xFF) / 100.0f;
    }

    return;
}

void testInnerStructHost(innerStruct *A, innerStruct *C, const int n)
{
    for (int idx = 0; idx < n; idx++)
    {
        C[idx].x = A[idx].x + 10.f;
        C[idx].y = A[idx].y + 20.f;
    }

    return;
}

void checkInnerStruct(innerStruct *hostRef, innerStruct *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i].x - gpuRef[i].x) > epsilon)
        {
            match = 0;
            printf("different on %dth element: host %f gpu %f\n", i,
                    hostRef[i].x, gpuRef[i].x);
            break;
        }

        if (abs(hostRef[i].y - gpuRef[i].y) > epsilon)
        {
            match = 0;
            printf("different on %dth element: host %f gpu %f\n", i,
                    hostRef[i].y, gpuRef[i].y);
            break;
        }
    }

    if (!match)  printf("Arrays do not match.\n\n");
}

__global__ void testInnerStruct(innerStruct *data, innerStruct * result,
                                const int n)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n)
    {
        innerStruct tmp = data[i];
        tmp.x += 10.f;
        tmp.y += 20.f;
        result[i] = tmp;
    }
}

__global__ void warmup(innerStruct *data, innerStruct * result, const int n)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n)
    {
        innerStruct tmp = data[i];
        tmp.x += 10.f;
        tmp.y += 20.f;
        result[i] = tmp;
    }
}

int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s test struct of array at ", argv[0]);
    printf("device %d: %s \n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // allocate host memory
    int nElem = LEN;
    size_t nBytes = nElem * sizeof(innerStruct);
    innerStruct     *h_A = (innerStruct *)malloc(nBytes);
    innerStruct *hostRef = (innerStruct *)malloc(nBytes);
    innerStruct *gpuRef  = (innerStruct *)malloc(nBytes);

    // initialize host array
    initialInnerStruct(h_A, nElem);
    testInnerStructHost(h_A, hostRef, nElem);

    // allocate device memory
    innerStruct *d_A, *d_C;
    CHECK(hipMalloc((innerStruct**)&d_A, nBytes));
    CHECK(hipMalloc((innerStruct**)&d_C, nBytes));

    // copy data from host to device
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));

    // set up offset for summaryAU: It is blocksize not offset. Thanks.CZ
    int blocksize = 128;

    if (argc > 1) blocksize = atoi(argv[1]);

    // execution configuration
    dim3 block (blocksize, 1);
    dim3 grid  ((nElem + block.x - 1) / block.x, 1);

    // kernel 1: warmup
    double iStart = seconds();
    warmup<<<grid, block>>>(d_A, d_C, nElem);
    CHECK(hipDeviceSynchronize());
    double iElaps = seconds() - iStart;
    printf("warmup      <<< %3d, %3d >>> elapsed %f sec\n", grid.x, block.x,
           iElaps);
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    checkInnerStruct(hostRef, gpuRef, nElem);
    CHECK(hipGetLastError());

    // kernel 2: testInnerStruct
    iStart = seconds();
    testInnerStruct<<<grid, block>>>(d_A, d_C, nElem);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("innerstruct <<< %3d, %3d >>> elapsed %f sec\n", grid.x, block.x,
           iElaps);
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    checkInnerStruct(hostRef, gpuRef, nElem);
    CHECK(hipGetLastError());

    // free memories both host and device
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_C));
    free(h_A);
    free(hostRef);
    free(gpuRef);

    // reset device
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}
