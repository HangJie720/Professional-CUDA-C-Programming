#include "../common/common.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>

/*
 * This is an example demonstrating usage of the cuSPARSE library to perform a
 * sparse matrix-vector multiplication on randomly generated data.
 */

/*
 * M = # of rows
 * N = # of columns
 */
int M = 1024;
int N = 1024;

/*
 * Generate a vector of length N with random single-precision floating-point
 * values between 0 and 100.
 */
void generate_random_vector(int N, float **outX)
{
    int i;
    double rMax = (double)RAND_MAX;
    float *X = (float *)malloc(sizeof(float) * N);

    for (i = 0; i < N; i++)
    {
        int r = rand();
        double dr = (double)r;
        X[i] = (dr / rMax) * 100.0;
    }

    *outX = X;
}

/*
 * Generate random dense matrix A in column-major order, while rounding some
 * elements down to zero to ensure it is sparse.
 */
int generate_random_dense_matrix(int M, int N, float **outA)
{
    int i, j;
    double rMax = (double)RAND_MAX;
    float *A = (float *)malloc(sizeof(float) * M * N);
    int totalNnz = 0;

    for (j = 0; j < N; j++)
    {
        for (i = 0; i < M; i++)
        {
            int r = rand();
            float *curr = A + (j * M + i);

            if (r % 3 > 0)
            {
                *curr = 0.0f;
            }
            else
            {
                double dr = (double)r;
                *curr = (dr / rMax) * 100.0;
            }

            if (*curr != 0.0f)
            {
                totalNnz++;
            }
        }
    }

    *outA = A;
    return totalNnz;
}

int main(int argc, char **argv)
{
    int row;
    float *A, *dA;
    int *dNnzPerRow;
    float *dCsrValA;
    int *dCsrRowPtrA;
    int *dCsrColIndA;
    int totalNnz;
    float alpha = 3.0f;
    float beta = 4.0f;
    float *dX, *X;
    float *dY, *Y;
    hipsparseHandle_t handle = 0;
    hipsparseMatDescr_t descr = 0;

    // Generate input
    srand(9384);
    int trueNnz = generate_random_dense_matrix(M, N, &A);
    generate_random_vector(N, &X);
    generate_random_vector(M, &Y);

    // Create the cuSPARSE handle
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    // Allocate device memory for vectors and the dense form of the matrix A
    CHECK(hipMalloc((void **)&dX, sizeof(float) * N));
    CHECK(hipMalloc((void **)&dY, sizeof(float) * M));
    CHECK(hipMalloc((void **)&dA, sizeof(float) * M * N));
    CHECK(hipMalloc((void **)&dNnzPerRow, sizeof(int) * M));

    // Construct a descriptor of the matrix A
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descr));
    CHECK_CUSPARSE(hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO));

    // Transfer the input vectors and dense matrix A to the device
    CHECK(hipMemcpy(dX, X, sizeof(float) * N, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dY, Y, sizeof(float) * M, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dA, A, sizeof(float) * M * N, hipMemcpyHostToDevice));

    // Compute the number of non-zero elements in A
    CHECK_CUSPARSE(hipsparseSnnz(handle, HIPSPARSE_DIRECTION_ROW, M, N, descr, dA,
                                M, dNnzPerRow, &totalNnz));

    if (totalNnz != trueNnz)
    {
        fprintf(stderr, "Difference detected between cuSPARSE NNZ and true "
                "value: expected %d but got %d\n", trueNnz, totalNnz);
        return 1;
    }

    // Allocate device memory to store the sparse CSR representation of A
    CHECK(hipMalloc((void **)&dCsrValA, sizeof(float) * totalNnz));
    CHECK(hipMalloc((void **)&dCsrRowPtrA, sizeof(int) * (M + 1)));
    CHECK(hipMalloc((void **)&dCsrColIndA, sizeof(int) * totalNnz));

    // Convert A from a dense formatting to a CSR formatting, using the GPU
    CHECK_CUSPARSE(hipsparseSdense2csr(handle, M, N, descr, dA, M, dNnzPerRow,
                                      dCsrValA, dCsrRowPtrA, dCsrColIndA));

    // Perform matrix-vector multiplication with the CSR-formatted matrix A
    CHECK_CUSPARSE(hipsparseScsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  M, N, totalNnz, &alpha, descr, dCsrValA,
                                  dCsrRowPtrA, dCsrColIndA, dX, &beta, dY));

    // Copy the result vector back to the host
    CHECK(hipMemcpy(Y, dY, sizeof(float) * M, hipMemcpyDeviceToHost));

    for (row = 0; row < 10; row++)
    {
        printf("%2.2f\n", Y[row]);
    }

    printf("...\n");

    free(A);
    free(X);
    free(Y);

    CHECK(hipFree(dX));
    CHECK(hipFree(dY));
    CHECK(hipFree(dA));
    CHECK(hipFree(dNnzPerRow));
    CHECK(hipFree(dCsrValA));
    CHECK(hipFree(dCsrRowPtrA));
    CHECK(hipFree(dCsrColIndA));

    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descr));
    CHECK_CUSPARSE(hipsparseDestroy(handle));


    return 0;
}
