#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

/*
 * This example uses the cuRAND host and device API to replace the system rand()
 * call by pre-generating large chunks of random numbers before fetching one at
 * at time. If there are no unused random numbers left, a new batch is generated
 * synchronously.
 */

/*
 * initialize_state initializes cuRAND device state
 */
__global__ void initialize_state(hiprandState *states)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(9384, tid, 0, states + tid);
}

/*
 * refill_randoms uses the cuRAND device API to generate N random values using
 * the states passed to the kernel.
 */
__global__ void refill_randoms(float *dRand, int N, hiprandState *states)
{
    int i;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int nthreads = gridDim.x * blockDim.x;
    hiprandState *state = states + tid;

    for (i = tid; i < N; i += nthreads)
    {
        dRand[i] = hiprand_uniform(state);
    }
}

/*
 * An implementation of rand() that uses the cuRAND device API.
 */
float cuda_device_rand()
{
    static hiprandState *states = NULL;
    static float *dRand = NULL;
    static float *hRand = NULL;
    static int dRand_length = 1000000;
    static int dRand_used = 1000000;

    int threads_per_block = 256;
    int blocks_per_grid = 30;

    if (dRand == NULL)
    {
        /*
         * If the cuRAND state hasn't been initialized yet, pre-allocate memory
         * to store the generated random values in as well as the cuRAND device
         * state objects.
         */
        CHECK(hipMalloc((void **)&dRand, sizeof(float) * dRand_length));
        CHECK(hipMalloc((void **)&states, sizeof(hiprandState) *
                        threads_per_block * blocks_per_grid));
        hRand = (float *)malloc(sizeof(float) * dRand_length);
        // Initialize states on the device
        initialize_state<<<blocks_per_grid, threads_per_block>>>(states);
    }

    if (dRand_used == dRand_length)
    {
        /*
         * If all pre-generated random numbers have been consumed, regenerate a
         * new batch.
         */
        refill_randoms<<<blocks_per_grid, threads_per_block>>>(dRand,
                dRand_length, states);
        CHECK(hipMemcpy(hRand, dRand, sizeof(float) * dRand_length,
                        hipMemcpyDeviceToHost));
        dRand_used = 0;
    }

    // Return the next pre-generated random number
    return hRand[dRand_used++];
}

/*
 * An implementation of rand() that uses the cuRAND host API.
 */
float cuda_host_rand()
{
    static float *dRand = NULL;
    static float *hRand = NULL;
    hiprandGenerator_t randGen;
    static int dRand_length = 1000000;
    static int dRand_used = 1000000;

    if (dRand == NULL)
    {
        /*
         * If the cuRAND state hasn't been initialized yet, construct a cuRAND
         * host generator and pre-allocate memory to store the generated random
         * values in.
         */
        CHECK_CURAND(hiprandCreateGenerator(&randGen,
                                           HIPRAND_RNG_PSEUDO_DEFAULT));
        CHECK(hipMalloc((void **)&dRand, sizeof(float) * dRand_length));
        hRand = (float *)malloc(sizeof(float) * dRand_length);
    }

    if (dRand_used == dRand_length)
    {
        /*
         * If all pre-generated random numbers have been consumed, regenerate a
         * new batch using hiprandGenerateUniform.
         */
        CHECK_CURAND(hiprandGenerateUniform(randGen, dRand, dRand_length));
        CHECK(hipMemcpy(hRand, dRand, sizeof(float) * dRand_length,
                        hipMemcpyDeviceToHost));
        dRand_used = 0;
    }

    // Return the next pre-generated random number
    return hRand[dRand_used++];
}

/*
 * A reference implementation that uses system rand().
 */
float host_rand()
{
    return (float)rand() / (float)RAND_MAX;
}

int main(int argc, char **argv)
{
    int i;
    int N = 8388608;

    /*
     * Allocate N random numbers from each of the random number generation
     * functions implemented.
     */
    for (i = 0; i < N; i++)
    {
        float h = host_rand();
        float d = cuda_host_rand();
        float dd = cuda_device_rand();
        printf("%2.4f %2.4f %2.4f\n", h, d, dd);
        getchar();
    }

    return 0;
}
