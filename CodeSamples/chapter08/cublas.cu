#include "../common/common.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

/*
 * A simple example of performing matrix-vector multiplication using the cuBLAS
 * library and some randomly generated inputs.
 */

/*
 * M = # of rows
 * N = # of columns
 */
int M = 1024;
int N = 1024;

/*
 * Generate a vector of length N with random single-precision floating-point
 * values between 0 and 100.
 */
void generate_random_vector(int N, float **outX)
{
    int i;
    double rMax = (double)RAND_MAX;
    float *X = (float *)malloc(sizeof(float) * N);

    for (i = 0; i < N; i++)
    {
        int r = rand();
        double dr = (double)r;
        X[i] = (dr / rMax) * 100.0;
    }

    *outX = X;
}

/*
 * Generate a matrix with M rows and N columns in column-major order. The matrix
 * will be filled with random single-precision floating-point values between 0
 * and 100.
 */
void generate_random_dense_matrix(int M, int N, float **outA)
{
    int i, j;
    double rMax = (double)RAND_MAX;
    float *A = (float *)malloc(sizeof(float) * M * N);

    // For each column
    for (j = 0; j < N; j++)
    {
        // For each row
        for (i = 0; i < M; i++)
        {
            double dr = (double)rand();
            A[j * M + i] = (dr / rMax) * 100.0;
        }
    }

    *outA = A;
}

int main(int argc, char **argv)
{
    int i;
    float *A, *dA;
    float *X, *dX;
    float *Y, *dY;
    float beta;
    float alpha;
    hipblasHandle_t handle = 0;

    alpha = 3.0f;
    beta = 4.0f;

    // Generate inputs
    srand(9384);
    generate_random_dense_matrix(M, N, &A);
    generate_random_vector(N, &X);
    generate_random_vector(M, &Y);

    // Create the cuBLAS handle
    CHECK_CUBLAS(hipblasCreate(&handle));

    // Allocate device memory
    CHECK(hipMalloc((void **)&dA, sizeof(float) * M * N));
    CHECK(hipMalloc((void **)&dX, sizeof(float) * N));
    CHECK(hipMalloc((void **)&dY, sizeof(float) * M));

    // Transfer inputs to the device
    CHECK_CUBLAS(hipblasSetVector(N, sizeof(float), X, 1, dX, 1));
    CHECK_CUBLAS(hipblasSetVector(M, sizeof(float), Y, 1, dY, 1));
    CHECK_CUBLAS(hipblasSetMatrix(M, N, sizeof(float), A, M, dA, M));

    // Execute the matrix-vector multiplication
    CHECK_CUBLAS(hipblasSgemv(handle, HIPBLAS_OP_N, M, N, &alpha, dA, M, dX, 1,
                             &beta, dY, 1));

    // Retrieve the output vector from the device
    CHECK_CUBLAS(hipblasGetVector(M, sizeof(float), dY, 1, Y, 1));

    for (i = 0; i < 10; i++)
    {
        printf("%2.2f\n", Y[i]);
    }

    printf("...\n");

    free(A);
    free(X);
    free(Y);

    CHECK(hipFree(dA));
    CHECK(hipFree(dY));
    CHECK_CUBLAS(hipblasDestroy(handle));

    return 0;
}
