#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <stdio.h>

/*
 * A simple introduction to programming in CUDA. This program prints "Hello
 * World from GPU! from 10 CUDA threads running on the GPU.
 */

__global__ void helloFromGPU()
{
    printf("Hello World from GPU!\n");
}

int main(int argc, char **argv)
{
    printf("Hello World from CPU!\n");

    helloFromGPU<<<1, 10>>>();
    CHECK(hipDeviceReset());
    return 0;
}


