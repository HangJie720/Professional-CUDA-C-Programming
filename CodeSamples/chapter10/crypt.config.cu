#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>

/*
 * The crypt application implements IDEA encryption and decryption of a single
 * input file using the secret key provided.
 */

// Chunking size for IDEA, in bytes
#define CHUNK_SIZE  8
// Length of the encryption/decryption keys, in bytes
#define KEY_LENGTH  52
#define BLOCK_SIZE_IN_CHUNKS    1024000
// Length of the secret key, in bytes
#define USERKEY_LENGTH  8
#define BITS_PER_BYTE   8

typedef enum { ENCRYPT, DECRYPT } action;

__constant__ int dkey[KEY_LENGTH];

/*
 * doCrypt implements the core logic of IDEA. It iterates over the byte
 * chunks stored in plainList and outputs their encrypted/decrypted form to the
 * corresponding element in cryptList using the secret key provided.
 */
__device__ void doCrypt(int chunk, signed char *plain, signed char *crypt)
{
    long x1, x2, x3, x4, t1, t2, ik, r;

    x1  = (((unsigned int)plain[chunk * CHUNK_SIZE]) & 0xff);
    x1 |= ((((unsigned int)plain[chunk * CHUNK_SIZE + 1]) & 0xff) <<
           BITS_PER_BYTE);
    x2  = (((unsigned int)plain[chunk * CHUNK_SIZE + 2]) & 0xff);
    x2 |= ((((unsigned int)plain[chunk * CHUNK_SIZE + 3]) & 0xff) <<
           BITS_PER_BYTE);
    x3  = (((unsigned int)plain[chunk * CHUNK_SIZE + 4]) & 0xff);
    x3 |= ((((unsigned int)plain[chunk * CHUNK_SIZE + 5]) & 0xff) <<
           BITS_PER_BYTE);
    x4  = (((unsigned int)plain[chunk * CHUNK_SIZE + 6]) & 0xff);
    x4 |= ((((unsigned int)plain[chunk * CHUNK_SIZE + 7]) & 0xff) <<
           BITS_PER_BYTE);
    ik  = 0;
    r = CHUNK_SIZE;

    do
    {
        x1 = (int)((((long)x1 * dkey[ik++]) % 0x10001L) & 0xffff);
        x2 = ((x2 + dkey[ik++]) & 0xffff);
        x3 = ((x3 + dkey[ik++]) & 0xffff);
        x4 = (int)((((long)x4 * dkey[ik++]) % 0x10001L) & 0xffff);

        t2 = (x1 ^ x3);
        t2 = (int)((((long)t2 * dkey[ik++]) % 0x10001L) & 0xffff);

        t1 = ((t2 + (x2 ^ x4)) & 0xffff);
        t1 = (int)((((long)t1 * dkey[ik++]) % 0x10001L) & 0xffff);
        t2 = (t1 + t2 & 0xffff);

        x1 = (x1 ^ t1);
        x4 = (x4 ^ t2);
        t2 = (t2 ^ x2);
        x2 = (x3 ^ t1);
        x3 = t2;
    }
    while(--r != 0);

    x1 = (int)((((long)x1 * dkey[ik++]) % 0x10001L) & 0xffff);
    x3 = ((x3 + dkey[ik++]) & 0xffff);
    x2 = ((x2 + dkey[ik++]) & 0xffff);
    x4 = (int)((((long)x4 * dkey[ik++]) % 0x10001L) & 0xffff);

    crypt[chunk * CHUNK_SIZE]     = (signed char) x1;
    crypt[chunk * CHUNK_SIZE + 1] = (signed char) ((unsigned long)x1 >>
                                    BITS_PER_BYTE);
    crypt[chunk * CHUNK_SIZE + 2] = (signed char) x3;
    crypt[chunk * CHUNK_SIZE + 3] = (signed char) ((unsigned long)x3 >>
                                    BITS_PER_BYTE);
    crypt[chunk * CHUNK_SIZE + 4] = (signed char) x2;
    crypt[chunk * CHUNK_SIZE + 5] = (signed char) ((unsigned long)x2 >>
                                    BITS_PER_BYTE);
    crypt[chunk * CHUNK_SIZE + 6] = (signed char) x4;
    crypt[chunk * CHUNK_SIZE + 7] = (signed char) ((unsigned long)x4 >>
                                    BITS_PER_BYTE);
}

__global__ void encrypt_decrypt(signed char *plain, signed char *crypt,
                                int nChunks)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int nthreads = blockDim.x * gridDim.x;

    for ( ; tid < nChunks; tid += nthreads)
    {
        doCrypt(tid, plain, crypt);
    }
}

static void encrypt_decrypt_driver(signed char *plain, signed char *crypt,
                                   int *key,
                                   int plainLength, int nThreadsPerBlock)
{
    hipEvent_t start, *finishes;
    hipStream_t *streams;
    int nChunks, b, nBlocks;
    signed char *dPlain, *dCrypt;

    if (plainLength % CHUNK_SIZE != 0)
    {
        fprintf(stderr, "Invalid encryption: length of plain must be an even "
                "multiple of %d but is %d\n", CHUNK_SIZE, plainLength);
        exit(-1);
    }

    hipDeviceProp_t info;
    CHECK(hipGetDeviceProperties(&info, 0));
    nChunks = plainLength / CHUNK_SIZE;
    nBlocks = (nChunks + BLOCK_SIZE_IN_CHUNKS - 1) / BLOCK_SIZE_IN_CHUNKS;
    int nThreadBlocks = (nChunks + nThreadsPerBlock - 1) / nThreadsPerBlock;

    if (nThreadBlocks > info.maxGridSize[0])
    {
        nThreadBlocks = info.maxGridSize[0];
    }

    CHECK(hipEventCreate(&start, 0));
    finishes = (hipEvent_t *)malloc(sizeof(hipEvent_t) * nBlocks);
    streams = (hipStream_t *)malloc(sizeof(hipStream_t) * nBlocks);

    for (b = 0; b < nBlocks; b++)
    {
        CHECK(hipStreamCreate(streams + b));
        CHECK(hipEventCreate(finishes + b));
    }

    CHECK(hipMalloc((void **)&dPlain,
                       plainLength * sizeof(signed char)));
    CHECK(hipMalloc((void **)&dCrypt,
                       plainLength * sizeof(signed char)));

    CHECK(hipEventRecord(start, streams[0]));
    CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(dkey), key, KEY_LENGTH * sizeof(int), 0,
                    hipMemcpyHostToDevice, streams[0]));
    CHECK(hipStreamSynchronize(streams[0]));

    for (b = 0; b < nBlocks; b++)
    {
        int blockOffset = b * BLOCK_SIZE_IN_CHUNKS * CHUNK_SIZE;
        int localChunks = BLOCK_SIZE_IN_CHUNKS;

        if (b * BLOCK_SIZE_IN_CHUNKS + localChunks > nChunks)
        {
            localChunks = nChunks - b * BLOCK_SIZE_IN_CHUNKS;
        }

        CHECK(hipMemcpyAsync(dPlain + blockOffset, plain + blockOffset,
                        localChunks * CHUNK_SIZE * sizeof(signed char),
                        hipMemcpyHostToDevice, streams[b]));

        encrypt_decrypt<<<nThreadBlocks, nThreadsPerBlock, 0, streams[b]>>>(
            dPlain + blockOffset, dCrypt + blockOffset, localChunks);
        CHECK(hipMemcpyAsync(crypt + blockOffset, dCrypt + blockOffset,
                        localChunks * CHUNK_SIZE * sizeof(signed char),
                        hipMemcpyDeviceToHost, streams[b]));
        CHECK(hipEventRecord(finishes[b], streams[b]));
    }

    CHECK(hipDeviceSynchronize());

    float maxElapsed = 0.0;

    for (b = 0; b < nBlocks; b++)
    {
        float elapsed;
        CHECK(hipEventElapsedTime(&elapsed, start, finishes[b]));
        maxElapsed = elapsed > maxElapsed ? elapsed : maxElapsed;
    }

    printf("Processed %d bytes in %f ms ( %f KB/ms )\n", plainLength,
           maxElapsed, ((float)plainLength / maxElapsed) / 1024.0f);

    for (b = 0; b < nBlocks; b++)
    {
        CHECK(hipStreamDestroy(streams[b]));
        CHECK(hipEventDestroy(finishes[b]));
    }

    free(streams);
    free(finishes);
    CHECK(hipEventDestroy(start));

    CHECK(hipFree(dPlain));
    CHECK(hipFree(dCrypt));
}

/*
 * Get the length of a file on disk.
 */
static size_t getFileLength(FILE *fp)
{
    fseek(fp, 0L, SEEK_END);
    size_t fileLen = ftell(fp);
    fseek(fp, 0L, SEEK_SET);
    return (fileLen);
}

/*
 * inv is used to generate the key used for decryption from the secret key.
 */
static int inv(int x)
{
    int t0, t1;
    int q, y;

    if (x <= 1)             // Assumes positive x.
        return (x);          // 0 and 1 are self-inverse.

    t1 = 0x10001 / x;       // (2**16+1)/x; x is >= 2, so fits 16 bits.
    y = 0x10001 % x;

    if (y == 1)
        return ((1 - t1) & 0xffff);

    t0 = 1;

    do
    {
        q = x / y;
        x = x % y;
        t0 += q * t1;

        if (x == 1) return (t0);

        q = y / x;
        y = y % x;
        t1 += q * t0;
    }
    while (y != 1);

    return ((1 - t1) & 0xffff);
}

/*
 * Generate the key to be used for encryption, based on the user key read from
 * disk.
 */
static int *generateEncryptKey(int16_t *userkey)
{
    int i, j;
    int *key;

    CHECK(hipHostMalloc(&key, KEY_LENGTH * sizeof(int)));
    memset(key, 0x00, sizeof(int) * KEY_LENGTH);

    for (i = 0; i < CHUNK_SIZE; i++)
    {
        key[i] = (userkey[i] & 0xffff);
    }

    for (i = CHUNK_SIZE; i < KEY_LENGTH; i++)
    {
        j = i % CHUNK_SIZE;

        if (j < 6)
        {
            key[i] = ((key[i - 7] >> 9) | (key[i - 6] << 7))
                     & 0xffff;
            continue;
        }

        if (j == 6)
        {
            key[i] = ((key[i - 7] >> 9) | (key[i - 14] << 7))
                     & 0xffff;
            continue;
        }

        key[i] = ((key[i - 15] >> 9) | (key[i - 14] << 7))
                 & 0xffff;
    }

    return (key);
}

/*
 * Generate the key to be used for decryption, based on the user key read from
 * disk.
 */
static int *generateDecryptKey(int16_t *userkey)
{
    int *key;
    int i, j, k;
    int t1, t2, t3;

    CHECK(hipHostMalloc(&key, KEY_LENGTH * sizeof(int)));
    int *Z = generateEncryptKey(userkey);

    t1 = inv(Z[0]);
    t2 = - Z[1] & 0xffff;
    t3 = - Z[2] & 0xffff;

    key[51] = inv(Z[3]);
    key[50] = t3;
    key[49] = t2;
    key[48] = t1;

    j = 47;
    k = 4;

    for (i = 0; i < 7; i++)
    {
        t1 = Z[k++];
        key[j--] = Z[k++];
        key[j--] = t1;
        t1 = inv(Z[k++]);
        t2 = -Z[k++] & 0xffff;
        t3 = -Z[k++] & 0xffff;
        key[j--] = inv(Z[k++]);
        key[j--] = t2;
        key[j--] = t3;
        key[j--] = t1;
    }

    t1 = Z[k++];
    key[j--] = Z[k++];
    key[j--] = t1;
    t1 = inv(Z[k++]);
    t2 = -Z[k++] & 0xffff;
    t3 = -Z[k++] & 0xffff;
    key[j--] = inv(Z[k++]);
    key[j--] = t3;
    key[j--] = t2;
    key[j--] = t1;

    CHECK(hipHostFree(Z));

    return (key);
}

void readInputData(FILE *in, size_t textLen, signed char **text,
                   signed char **crypt)
{
    CHECK(hipHostMalloc(text, textLen * sizeof(signed char)));
    CHECK(hipHostMalloc(crypt, textLen * sizeof(signed char)));

    if (fread(*text, sizeof(signed char), textLen, in) != textLen)
    {
        fprintf(stderr, "Failed reading text from input file\n");
        exit(1);
    }
}

void cleanup(signed char *text, signed char *crypt, int *key,
             int16_t *userkey)
{
    free(userkey);
    CHECK(hipHostFree(key));
    CHECK(hipHostFree(text));
    CHECK(hipHostFree(crypt));
}

/*
 * Initialize application state by reading inputs from the disk and
 * pre-allocating memory. Hand off to encrypt_decrypt to perform the actualy
 * encryption or decryption. Then, write the encrypted/decrypted results to
 * disk.
 */
int main(int argc, char **argv)
{
    FILE *in, *out, *keyfile;
    signed char *text, *crypt;
    size_t textLen, keyFileLength;
    int16_t *userkey;
    int *key;
    action a;

    if (argc != 6)
    {
        printf("usage: %s <encrypt|decrypt> <file.in> <file.out> <key.file> "
               "<threads-per-block>\n", argv[0]);
        return (1);
    }

    // Are we encrypting or decrypting?
    if (strncmp(argv[1], "encrypt", 7) == 0)
    {
        a = ENCRYPT;
    }
    else if (strncmp(argv[1], "decrypt", 7) == 0)
    {
        a = DECRYPT;
    }
    else
    {
        fprintf(stderr, "The action specified ('%s') is not valid. Must be "
                "either 'encrypt' or 'decrypt'\n", argv[1]);
        return (1);
    }

    // Input file
    in = fopen(argv[2], "r");

    if (in == NULL)
    {
        fprintf(stderr, "Unable to open %s for reading\n", argv[2]);
        return (1);
    }

    // Output file
    out = fopen(argv[3], "w");

    if (out == NULL)
    {
        fprintf(stderr, "Unable to open %s for writing\n", argv[3]);
        return (1);
    }

    // Key file
    keyfile = fopen(argv[4], "r");

    if (keyfile == NULL)
    {
        fprintf(stderr, "Unable to open key file %s for reading\n", argv[4]);
        return (1);
    }

    int nThreadsPerBlock = atoi(argv[5]);

    keyFileLength = getFileLength(keyfile);

    if (keyFileLength != sizeof(*userkey) * USERKEY_LENGTH)
    {
        fprintf(stderr, "Invalid user key file length %lu, must be %lu\n",
                keyFileLength, sizeof(*userkey) * USERKEY_LENGTH);
        return (1);
    }

    userkey = (int16_t *)malloc(sizeof(int16_t) * USERKEY_LENGTH);

    if (userkey == NULL)
    {
        fprintf(stderr, "Error allocating user key\n");
        return (1);
    }

    if (fread(userkey, sizeof(*userkey), USERKEY_LENGTH, keyfile) !=
            USERKEY_LENGTH)
    {
        fprintf(stderr, "Error reading user key\n");
        return (1);
    }

    if (a == ENCRYPT)
    {
        key = generateEncryptKey(userkey);
    }
    else
    {
        key = generateDecryptKey(userkey);
    }

    textLen = getFileLength(in);

    if (textLen % CHUNK_SIZE != 0)
    {
        fprintf(stderr, "Invalid input file length %lu, must be evenly "
                "divisible by %d\n", textLen, CHUNK_SIZE);
        return (1);
    }

    readInputData(in, textLen, &text, &crypt);
    fclose(in);

    encrypt_decrypt_driver(text, crypt, key, textLen, nThreadsPerBlock);

    if (fwrite(crypt, sizeof(signed char), textLen, out) != textLen)
    {
        fprintf(stderr, "Failed writing crypt to %s\n", argv[3]);
        return (1);
    }

    fclose(out);

    cleanup(text, crypt, key, userkey);

    return (0);
}
