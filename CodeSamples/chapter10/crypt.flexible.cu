#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>

/*
 * The crypt application implements IDEA encryption and decryption of a single
 * input file using the secret key provided.
 */

// Chunking size for IDEA, in bytes
#define CHUNK_SIZE  8
// Length of the encryption/decryption keys, in bytes
#define KEY_LENGTH  52
#define BLOCK_SIZE_IN_CHUNKS    1024000
// Length of the secret key, in bytes
#define USERKEY_LENGTH  8
#define BITS_PER_BYTE   8

typedef struct _device_context
{
    signed char *dPlain, *dCrypt;
    hipStream_t *streams;
    int nBlocks;
} device_context;

typedef enum { ENCRYPT, DECRYPT } action;

__constant__ int dkey[KEY_LENGTH];

/*
 * doCrypt implements the core logic of IDEA. It iterates over the byte
 * chunks stored in plainList and outputs their encrypted/decrypted form to the
 * corresponding element in cryptList using the secret key provided.
 */
__host__ __device__ void doCrypt(int chunk, signed char *plain,
                                 signed char *crypt, int *key)
{
    long x1, x2, x3, x4, t1, t2, ik, r;

    x1  = (((unsigned int)plain[chunk * CHUNK_SIZE]) & 0xff);
    x1 |= ((((unsigned int)plain[chunk * CHUNK_SIZE + 1]) & 0xff) <<
           BITS_PER_BYTE);
    x2  = (((unsigned int)plain[chunk * CHUNK_SIZE + 2]) & 0xff);
    x2 |= ((((unsigned int)plain[chunk * CHUNK_SIZE + 3]) & 0xff) <<
           BITS_PER_BYTE);
    x3  = (((unsigned int)plain[chunk * CHUNK_SIZE + 4]) & 0xff);
    x3 |= ((((unsigned int)plain[chunk * CHUNK_SIZE + 5]) & 0xff) <<
           BITS_PER_BYTE);
    x4  = (((unsigned int)plain[chunk * CHUNK_SIZE + 6]) & 0xff);
    x4 |= ((((unsigned int)plain[chunk * CHUNK_SIZE + 7]) & 0xff) <<
           BITS_PER_BYTE);
    ik  = 0;
    r = CHUNK_SIZE;

    do
    {
        x1 = (int)((((long)x1 * key[ik++]) % 0x10001L) & 0xffff);
        x2 = ((x2 + key[ik++]) & 0xffff);
        x3 = ((x3 + key[ik++]) & 0xffff);
        x4 = (int)((((long)x4 * key[ik++]) % 0x10001L) & 0xffff);

        t2 = (x1 ^ x3);
        t2 = (int)((((long)t2 * key[ik++]) % 0x10001L) & 0xffff);

        t1 = ((t2 + (x2 ^ x4)) & 0xffff);
        t1 = (int)((((long)t1 * key[ik++]) % 0x10001L) & 0xffff);
        t2 = (t1 + t2 & 0xffff);

        x1 = (x1 ^ t1);
        x4 = (x4 ^ t2);
        t2 = (t2 ^ x2);
        x2 = (x3 ^ t1);
        x3 = t2;
    }
    while(--r != 0);

    x1 = (int)((((long)x1 * key[ik++]) % 0x10001L) & 0xffff);
    x3 = ((x3 + key[ik++]) & 0xffff);
    x2 = ((x2 + key[ik++]) & 0xffff);
    x4 = (int)((((long)x4 * key[ik++]) % 0x10001L) & 0xffff);

    crypt[chunk * CHUNK_SIZE]     = (signed char) x1;
    crypt[chunk * CHUNK_SIZE + 1] = (signed char) ((unsigned long)x1 >>
                                    BITS_PER_BYTE);
    crypt[chunk * CHUNK_SIZE + 2] = (signed char) x3;
    crypt[chunk * CHUNK_SIZE + 3] = (signed char) ((unsigned long)x3 >>
                                    BITS_PER_BYTE);
    crypt[chunk * CHUNK_SIZE + 4] = (signed char) x2;
    crypt[chunk * CHUNK_SIZE + 5] = (signed char) ((unsigned long)x2 >>
                                    BITS_PER_BYTE);
    crypt[chunk * CHUNK_SIZE + 6] = (signed char) x4;
    crypt[chunk * CHUNK_SIZE + 7] = (signed char) ((unsigned long)x4 >>
                                    BITS_PER_BYTE);
}

__global__ void d_encrypt_decrypt(signed char *plain, signed char *crypt,
                                  int nChunks)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int nthreads = blockDim.x * gridDim.x;

    for ( ; tid < nChunks; tid += nthreads)
    {
        doCrypt(tid, plain, crypt, dkey);
    }
}

static void h_encrypt_decrypt(signed char *plain, signed char *crypt, int *key,
                              int plainLength)
{
    int c;
    int nChunks = plainLength / CHUNK_SIZE;

    for (c = 0; c < nChunks; c++)
    {
        doCrypt(c, plain, crypt, key);
    }
}

static void init_context(device_context *ctx, int plainLength)
{
    signed char *dPlain, *dCrypt;
    hipStream_t *streams;
    int nBlocks, b;

    if (plainLength % CHUNK_SIZE != 0)
    {
        fprintf(stderr, "Invalid encryption: length of plain must be an even "
                "multiple of %d but is %d\n", CHUNK_SIZE, plainLength);
        exit(-1);
    }

    CHECK(hipMalloc((void **)&dPlain,
                       plainLength * sizeof(signed char)));
    CHECK(hipMalloc((void **)&dCrypt,
                       plainLength * sizeof(signed char)));

    int nChunks = plainLength / CHUNK_SIZE;
    nBlocks = (nChunks + BLOCK_SIZE_IN_CHUNKS - 1) / BLOCK_SIZE_IN_CHUNKS;

    streams = (hipStream_t *)malloc(sizeof(hipStream_t) * nBlocks);

    for (b = 0; b < nBlocks; b++)
    {
        CHECK(hipStreamCreate(streams + b));
    }

    ctx->dPlain = dPlain;
    ctx->dCrypt = dCrypt;
    ctx->streams = streams;
    ctx->nBlocks = nBlocks;
}

static void encrypt_decrypt_driver(signed char *plain, signed char *crypt,
                                   int *key,
                                   int plainLength, int nThreadsPerBlock,
                                   device_context *ctx)
{
    int b;

    hipDeviceProp_t info;
    CHECK(hipGetDeviceProperties(&info, 0));
    int nChunks = plainLength / CHUNK_SIZE;
    int nThreadBlocks = (nChunks + nThreadsPerBlock - 1) / nThreadsPerBlock;

    if (nThreadBlocks > info.maxGridSize[0])
    {
        nThreadBlocks = info.maxGridSize[0];
    }

    CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(dkey), key, KEY_LENGTH * sizeof(int), 0,
                hipMemcpyHostToDevice, (ctx->streams)[0]));
    CHECK(hipStreamSynchronize((ctx->streams)[0]));

    for (b = 0; b < ctx->nBlocks; b++)
    {
        int blockOffset = b * BLOCK_SIZE_IN_CHUNKS * CHUNK_SIZE;
        int localChunks = BLOCK_SIZE_IN_CHUNKS;

        if (b * BLOCK_SIZE_IN_CHUNKS + localChunks > nChunks)
        {
            localChunks = nChunks - b * BLOCK_SIZE_IN_CHUNKS;
        }

        CHECK(hipMemcpyAsync(ctx->dPlain + blockOffset, plain + blockOffset,
                    localChunks * CHUNK_SIZE * sizeof(signed char),
                    hipMemcpyHostToDevice, (ctx->streams)[b]));

        d_encrypt_decrypt<<<nThreadBlocks, nThreadsPerBlock, 0,
            (ctx->streams)[b]>>>(ctx->dPlain + blockOffset,
                    ctx->dCrypt + blockOffset, localChunks);
        CHECK(hipMemcpyAsync(crypt + blockOffset, ctx->dCrypt + blockOffset,
                    localChunks * CHUNK_SIZE * sizeof(signed char),
                    hipMemcpyDeviceToHost, (ctx->streams)[b]));
    }
}

static void cleanup_context(device_context *ctx)
{
    int b;

    for (b = 0; b < ctx->nBlocks; b++)
    {
        CHECK(hipStreamDestroy(ctx->streams[b]));
    }

    free(ctx->streams);

    CHECK(hipFree(ctx->dPlain));
    CHECK(hipFree(ctx->dCrypt));
}

/*
 * Get the length of a file on disk.
 */
static size_t getFileLength(FILE *fp)
{
    fseek(fp, 0L, SEEK_END);
    size_t fileLen = ftell(fp);
    fseek(fp, 0L, SEEK_SET);
    return (fileLen);
}

/*
 * inv is used to generate the key used for decryption from the secret key.
 */
static int inv(int x)
{
    int t0, t1;
    int q, y;

    if (x <= 1)             // Assumes positive x.
        return (x);          // 0 and 1 are self-inverse.

    t1 = 0x10001 / x;       // (2**16+1)/x; x is >= 2, so fits 16 bits.
    y = 0x10001 % x;

    if (y == 1)
        return ((1 - t1) & 0xffff);

    t0 = 1;

    do
    {
        q = x / y;
        x = x % y;
        t0 += q * t1;

        if (x == 1) return (t0);

        q = y / x;
        y = y % x;
        t1 += q * t0;
    }
    while (y != 1);

    return ((1 - t1) & 0xffff);
}

/*
 * Generate the key to be used for encryption, based on the user key read from
 * disk.
 */
static int *generateEncryptKey(int16_t *userkey)
{
    int i, j;
    int *key;

    CHECK(hipHostMalloc(&key, KEY_LENGTH * sizeof(int)));
    memset(key, 0x00, sizeof(int) * KEY_LENGTH);

    for (i = 0; i < CHUNK_SIZE; i++)
    {
        key[i] = (userkey[i] & 0xffff);
    }

    for (i = CHUNK_SIZE; i < KEY_LENGTH; i++)
    {
        j = i % CHUNK_SIZE;

        if (j < 6)
        {
            key[i] = ((key[i - 7] >> 9) | (key[i - 6] << 7))
                     & 0xffff;
            continue;
        }

        if (j == 6)
        {
            key[i] = ((key[i - 7] >> 9) | (key[i - 14] << 7))
                     & 0xffff;
            continue;
        }

        key[i] = ((key[i - 15] >> 9) | (key[i - 14] << 7))
                 & 0xffff;
    }

    return (key);
}

/*
 * Generate the key to be used for decryption, based on the user key read from
 * disk.
 */
static int *generateDecryptKey(int16_t *userkey)
{
    int *key;
    int i, j, k;
    int t1, t2, t3;

    CHECK(hipHostMalloc(&key, KEY_LENGTH * sizeof(int)));
    int *Z = generateEncryptKey(userkey);

    t1 = inv(Z[0]);
    t2 = - Z[1] & 0xffff;
    t3 = - Z[2] & 0xffff;

    key[51] = inv(Z[3]);
    key[50] = t3;
    key[49] = t2;
    key[48] = t1;

    j = 47;
    k = 4;

    for (i = 0; i < 7; i++)
    {
        t1 = Z[k++];
        key[j--] = Z[k++];
        key[j--] = t1;
        t1 = inv(Z[k++]);
        t2 = -Z[k++] & 0xffff;
        t3 = -Z[k++] & 0xffff;
        key[j--] = inv(Z[k++]);
        key[j--] = t2;
        key[j--] = t3;
        key[j--] = t1;
    }

    t1 = Z[k++];
    key[j--] = Z[k++];
    key[j--] = t1;
    t1 = inv(Z[k++]);
    t2 = -Z[k++] & 0xffff;
    t3 = -Z[k++] & 0xffff;
    key[j--] = inv(Z[k++]);
    key[j--] = t3;
    key[j--] = t2;
    key[j--] = t1;

    CHECK(hipHostFree(Z));

    return (key);
}

void readInputData(FILE *in, size_t textLen, signed char **text,
                   signed char **crypt)
{
    CHECK(hipHostMalloc(text, textLen * sizeof(signed char)));
    CHECK(hipHostMalloc(crypt, textLen * sizeof(signed char)));

    if (fread(*text, sizeof(signed char), textLen, in) != textLen)
    {
        fprintf(stderr, "Failed reading text from input file\n");
        exit(1);
    }
}

void cleanup(signed char *text, signed char *crypt, int *key,
             int16_t *userkey)
{
    free(userkey);
    CHECK(hipHostFree(key));
    CHECK(hipHostFree(text));
    CHECK(hipHostFree(crypt));
}

/*
 * Initialize application state by reading inputs from the disk and
 * pre-allocating memory. Hand off to encrypt_decrypt to perform the actualy
 * encryption or decryption. Then, write the encrypted/decrypted results to
 * disk.
 */
int main(int argc, char **argv)
{
    FILE *in, *out, *keyfile;
    signed char *text, *crypt;
    size_t textLen, keyFileLength;
    int16_t *userkey;
    int *key;
    action a;
    hipEvent_t startEvent, finishEvent;

    if (argc != 6)
    {
        printf("usage: %s <encrypt|decrypt> <file.in> <file.out> <key.file> "
               "<threads-per-block>\n", argv[0]);
        return (1);
    }

    // Are we encrypting or decrypting?
    if (strncmp(argv[1], "encrypt", 7) == 0)
    {
        a = ENCRYPT;
    }
    else if (strncmp(argv[1], "decrypt", 7) == 0)
    {
        a = DECRYPT;
    }
    else
    {
        fprintf(stderr, "The action specified ('%s') is not valid. Must be "
                "either 'encrypt' or 'decrypt'\n", argv[1]);
        return (1);
    }

    // Input file
    in = fopen(argv[2], "r");

    if (in == NULL)
    {
        fprintf(stderr, "Unable to open %s for reading\n", argv[2]);
        return (1);
    }

    // Output file
    out = fopen(argv[3], "w");

    if (out == NULL)
    {
        fprintf(stderr, "Unable to open %s for writing\n", argv[3]);
        return (1);
    }

    // Key file
    keyfile = fopen(argv[4], "r");

    if (keyfile == NULL)
    {
        fprintf(stderr, "Unable to open key file %s for reading\n", argv[4]);
        return (1);
    }

    int nThreadsPerBlock = atoi(argv[5]);

    keyFileLength = getFileLength(keyfile);

    if (keyFileLength != sizeof(*userkey) * USERKEY_LENGTH)
    {
        fprintf(stderr, "Invalid user key file length %lu, must be %lu\n",
                keyFileLength, sizeof(*userkey) * USERKEY_LENGTH);
        return (1);
    }

    userkey = (int16_t *)malloc(sizeof(int16_t) * USERKEY_LENGTH);

    if (userkey == NULL)
    {
        fprintf(stderr, "Error allocating user key\n");
        return (1);
    }

    if (fread(userkey, sizeof(*userkey), USERKEY_LENGTH, keyfile) !=
            USERKEY_LENGTH)
    {
        fprintf(stderr, "Error reading user key\n");
        return (1);
    }

    if (a == ENCRYPT)
    {
        key = generateEncryptKey(userkey);
    }
    else
    {
        key = generateDecryptKey(userkey);
    }

    textLen = getFileLength(in);

    if (textLen % CHUNK_SIZE != 0)
    {
        fprintf(stderr, "Invalid input file length %lu, must be evenly "
                "divisible by %d\n", textLen, CHUNK_SIZE);
        return (1);
    }

    readInputData(in, textLen, &text, &crypt);
    fclose(in);

    int nDevices;

    if (hipGetDeviceCount(&nDevices) == hipErrorNoDevice) {
        // If no devices are found, run all computation on the CPU.
        double overall_start = seconds();
        h_encrypt_decrypt(text, crypt, key, textLen);
        double overall_finish = seconds();
        double overall_ms = 1000.0 * (overall_finish - overall_start);
        printf("Processed %d bytes in %.3f s on CPU ( %.4f KB/ms )\n",
                textLen, overall_ms,
                ((float)textLen / overall_ms) / 1024.0f);
    }
    else
    {
        int d;

        int nTotalChunks = textLen / CHUNK_SIZE;
        int chunksPerDevice = (nTotalChunks + nDevices - 1) / nDevices;

        device_context *ctxs = (device_context *)malloc(nDevices *
                sizeof(device_context));

        for (d = 0; d < nDevices; d++)
        {
            CHECK(hipSetDevice(d));
            int start = d * chunksPerDevice * CHUNK_SIZE;
            int len = chunksPerDevice * CHUNK_SIZE;

            if (start + len > textLen)
            {
                len = textLen - start;
            }

            init_context(ctxs + d, len);
        }

        CHECK(hipEventCreate(&startEvent));
        CHECK(hipEventCreate(&finishEvent));

        /*
         * Iterate over each device, launching a subset of the total chunks at
         * a time.
         */
        double overall_start = seconds();
        CHECK(hipEventRecord(startEvent));

        for (d = 0; d < nDevices; d++)
        {
            CHECK(hipSetDevice(d));
            int start = d * chunksPerDevice * CHUNK_SIZE;
            int len = chunksPerDevice * CHUNK_SIZE;

            if (start + len > textLen)
            {
                len = textLen - start;
            }

            encrypt_decrypt_driver(text + start, crypt + start, key, len,
                    nThreadsPerBlock, ctxs + d);
        }

        CHECK(hipEventRecord(finishEvent));

        // Wait for each device to finish its work.
        for (d = 0; d < nDevices; d++)
        {
            CHECK(hipSetDevice(d));
            CHECK(hipDeviceSynchronize());
        }

        double overall_finish = seconds();

        for (d = 0; d < nDevices; d++)
        {
            // Clean up any CUDA resource allocated for this device.
            CHECK(hipSetDevice(d));
            cleanup_context(ctxs + d);
        }

        float gpuElapsed;
        CHECK(hipEventElapsedTime(&gpuElapsed, startEvent, finishEvent));
        printf("Processed %d bytes in %.3f ms on GPUs ( %.4f KB/ms )\n",
                textLen, gpuElapsed, ((float)textLen / gpuElapsed) / 1024.0f);
        // Display the aggregate performance of all devices.
        double overall_elapsed_ms = 1000.0 * (overall_finish - overall_start);
        printf("In total, processed %d bytes in %.3f ms on %d devices\n",
                textLen, overall_elapsed_ms, nDevices);
        printf("Aggregate bandwith = %.4f KB/ms\n",
                (float)(textLen / 1024) / overall_elapsed_ms);
        free(ctxs);
    }

    if (fwrite(crypt, sizeof(signed char), textLen, out) != textLen)
    {
        fprintf(stderr, "Failed writing crypt to %s\n", argv[3]);
        return (1);
    }

    fclose(out);

    cleanup(text, crypt, key, userkey);

    return (0);
}
