#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <stdio.h>
#include <stdlib.h>

/**
 * This example illustrates different approaches to optimizing access to a
 * single shared variable by limiting conflicting, atomic operations on it.
 *
 * The first kernel, naive_reduction, simply performs an atomicAdd from every
 * thread on the same shared variable.
 *
 * simple_reduction first stores the values to be added together in shared
 * memory. Then, a single thread iterates over those values and computes a
 * partial sum. Finally, that partial sum is added to the global result using an
 * atomicAdd.
 *
 * parallel_reduction is the most complex example. It performs a parallel
 * reduction within each thread block. The partial result produced by that
 * local reduction is then added to the global result with an atomicAdd.
 *
 * The core of each of these kernels is wrapped in a loop to augment the amount
 * of work done and make timing the kernels at the millisecond granularity
 * feasible.
 **/

/**
 * This implementation makes use of shared memory and local reduction to improve
 * performance and decrease contention
 **/
__global__ void simple_reduction(int *shared_var, int *input_values, int N,
                                 int iters)
{
    __shared__ int local_mem[256];
    int iter, i;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int local_tid = threadIdx.x;
    int local_dim = blockDim.x;
    int minThreadInThisBlock = blockIdx.x * blockDim.x;
    int maxThreadInThisBlock = minThreadInThisBlock + (blockDim.x - 1);

    if (maxThreadInThisBlock >= N)
    {
        local_dim = N - minThreadInThisBlock;
    }

    for (iter = 0; iter < iters; iter++)
    {
        if (tid < N)
        {
            local_mem[local_tid] = input_values[tid];
        }

        // Required for correctness
        // __syncthreads();

        /*
         * Perform the local reduction across values written to shared memory
         * by threads in this thread block.
         */
        if (local_tid == 0)
        {
            int sum = 0;

            for (i = 0; i < local_dim; i++)
            {
                sum = sum + local_mem[i];
            }

            atomicAdd(shared_var, sum);
        }

        // Required for correctness
        // __syncthreads();
    }
}

int main(int argc, char **argv)
{
    int N = 20480;
    int block = 256;
    int device_iters = 3;
    int runs = 1;
    int i, true_value;
    int *d_shared_var, *d_input_values, *h_input_values;
    int h_sum;
    double mean_time = 0.0;

    CHECK(hipMalloc((void **)&d_shared_var, sizeof(int)));
    CHECK(hipMalloc((void **)&d_input_values, N * sizeof(int)));
    h_input_values = (int *)malloc(N * sizeof(int));

    for (i = 0; i < N; i++)
    {
        h_input_values[i] = i;
        true_value += i;
    }

    true_value *= device_iters;

    for (i = 0; i < runs; i++)
    {
        CHECK(hipMemset(d_shared_var, 0x00, sizeof(int)));
        CHECK(hipMemcpy(d_input_values, h_input_values, N * sizeof(int),
                         hipMemcpyHostToDevice));
        double start = seconds();

        simple_reduction<<<N / block, block>>>(d_shared_var,
                d_input_values, N, device_iters);

        CHECK(hipDeviceSynchronize());
        mean_time += seconds() - start;
        CHECK(hipMemcpy(&h_sum, d_shared_var, sizeof(int),
                         hipMemcpyDeviceToHost));

        if (h_sum != true_value)
        {
            fprintf(stderr, "Validation failure: expected %d, got %d\n",
                    true_value, h_sum);
            return 1;
        }
    }

    mean_time /= runs;

    printf("Mean execution time for reduction: %.4f ms\n",
           mean_time * 1000.0);

    return 0;
}
