#include "../common/common.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

/*
 * Fetches basic information on the first device in the current CUDA platform,
 * including number of SMs, bytes of constant memory, bytes of shared memory per
 * block, etc.
 */

int main(int argc, char *argv[])
{
    int iDev = 0;
    hipDeviceProp_t iProp;
    CHECK(hipGetDeviceProperties(&iProp, iDev));

    printf("Device %d: %s\n", iDev, iProp.name);
    printf("  Number of multiprocessors:                     %d\n",
           iProp.multiProcessorCount);
    printf("  Total amount of constant memory:               %4.2f KB\n",
           iProp.totalConstMem / 1024.0);
    printf("  Total amount of shared memory per block:       %4.2f KB\n",
           iProp.sharedMemPerBlock / 1024.0);
    printf("  Total number of registers available per block: %d\n",
           iProp.regsPerBlock);
    printf("  Warp size:                                     %d\n",
           iProp.warpSize);
    printf("  Maximum number of threads per block:           %d\n",
           iProp.maxThreadsPerBlock);
    printf("  Maximum number of threads per multiprocessor:  %d\n",
           iProp.maxThreadsPerMultiProcessor);
    printf("  Maximum number of warps per multiprocessor:    %d\n",
           iProp.maxThreadsPerMultiProcessor / 32);
    return EXIT_SUCCESS;
}
