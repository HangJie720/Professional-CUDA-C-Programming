#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#define LOG 0

/*
 * An implementation of parallel reduction using nested kernel launches from
 * CUDA kernels. This version adds optimizations on to the work in
 * nestedReduce.cu.
 */

// Recursive Implementation of Interleaved Pair Approach
int cpuRecursiveReduce(int *data, int const size)
{
    // stop condition
    if (size == 1) return data[0];

    // renew the stride
    int const stride = size / 2;

    // in-place reduction
    for (int i = 0; i < stride; i++)
    {
        data[i] += data[i + stride];
    }

    // call recursively
    return cpuRecursiveReduce(data, stride);
}

// Neighbored Pair Implementation with divergence
__global__ void reduceNeighbored (int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
    if (idx >= n) return;

    // in-place reduction in global memory
    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        if ((tid % (2 * stride)) == 0)
        {
            idata[tid] += idata[tid + stride];
        }

        // synchronize within threadblock
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

__global__ void gpuRecursiveReduce (int *g_idata, int *g_odata,
                                    unsigned int isize)
{
    // set thread ID
    unsigned int tid = threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;
    int *odata = &g_odata[blockIdx.x];

    // stop condition
    if (isize == 2 && tid == 0)
    {
        g_odata[blockIdx.x] = idata[0] + idata[1];
        return;
    }

    // nested invocation
    int istride = isize >> 1;

    if(istride > 1 && tid < istride)
    {
        // in place reduction
        idata[tid] += idata[tid + istride];
    }

    // sync at block level
    __syncthreads();

    // nested invocation to generate child grids
    if(tid == 0)
    {
        gpuRecursiveReduce<<<1, istride>>>(idata, odata, istride);

        // sync all child grids launched in this block
        hipDeviceSynchronize();
    }

    // sync at block level again
    __syncthreads();
}

__global__ void gpuRecursiveReduceNosync (int *g_idata, int *g_odata,
        unsigned int isize)
{
    // set thread ID
    unsigned int tid = threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;
    int *odata = &g_odata[blockIdx.x];

    // stop condition
    if (isize == 2 && tid == 0)
    {
        g_odata[blockIdx.x] = idata[0] + idata[1];
        return;
    }

    // nested invoke
    int istride = isize >> 1;

    if(istride > 1 && tid < istride)
    {
        idata[tid] += idata[tid + istride];

        if(tid == 0)
        {
            gpuRecursiveReduceNosync<<<1, istride>>>(idata, odata, istride);
        }
    }
}

// main from here
int main(int argc, char **argv)
{
    // set up device
    int dev = 0, gpu_sum;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s starting reduction at ", argv[0]);
    printf("device %d: %s ", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    bool bResult = false;

    // set up execution configuration
    int nblock  = 2048;
    int nthread = 512;   // initial block size

    if(argc > 1)
    {
        nblock = atoi(argv[1]);   // block size from command line argument
    }

    if(argc > 2)
    {
        nthread = atoi(argv[2]);   // block size from command line argument
    }

    int size = nblock * nthread; // total number of elements to reduceNeighbored

    dim3 block (nthread, 1);
    dim3 grid  ((size + block.x - 1) / block.x, 1);
    printf("array %d grid %d block %d\n", size, grid.x, block.x);

    // allocate host memory
    size_t bytes = size * sizeof(int);
    int *h_idata = (int *) malloc(bytes);
    int *h_odata = (int *) malloc(grid.x * sizeof(int));
    int *tmp     = (int *) malloc(bytes);

    // initialize the array
    for (int i = 0; i < size; i++)
    {
        h_idata[i] = (int)( rand() & 0xFF );
        h_idata[i] = 1;
    }

    memcpy (tmp, h_idata, bytes);

    // allocate device memory
    int *d_idata = NULL;
    int *d_odata = NULL;
    CHECK(hipMalloc((void **) &d_idata, bytes));
    CHECK(hipMalloc((void **) &d_odata, grid.x * sizeof(int)));

    double iStart, iElaps;

    // cpu recursive reduction
    iStart = seconds();
    int cpu_sum = cpuRecursiveReduce (tmp, size);
    iElaps = seconds() - iStart;
    printf("cpu reduce\t\telapsed %f sec cpu_sum: %d\n", iElaps, cpu_sum);

    // gpu reduceNeighbored
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    iStart = seconds();
    reduceNeighbored<<<grid, block>>>(d_idata, d_odata, size);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());
    iElaps = seconds() - iStart;
    CHECK(hipMemcpy(h_odata, d_odata, grid.x * sizeof(int),
                     hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];

    printf("gpu Neighbored\t\telapsed %f sec gpu_sum: %d <<<grid %d block "
           "%d>>>\n", iElaps, gpu_sum, grid.x, block.x);

    // gpu nested reduce kernel
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    iStart = seconds();
    gpuRecursiveReduce<<<grid, block>>>(d_idata, d_odata, block.x);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());
    iElaps = seconds() - iStart;
    CHECK(hipMemcpy(h_odata, d_odata, grid.x * sizeof(int),
                     hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];

    printf("gpu nested\t\telapsed %f sec gpu_sum: %d <<<grid %d block %d>>>\n",
           iElaps, gpu_sum, grid.x, block.x);

    // gpu nested reduce kernel without synchronization
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    iStart = seconds();
    gpuRecursiveReduceNosync<<<grid, block>>>(d_idata, d_odata, block.x);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());
    iElaps = seconds() - iStart;
    CHECK(hipMemcpy(h_odata, d_odata, grid.x * sizeof(int),
                     hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];

    printf("gpu nestedNosyn\t\telapsed %f sec gpu_sum: %d <<<grid %d block "
           "%d>>>\n", iElaps, gpu_sum, grid.x, block.x);

    // free host memory
    free(h_idata);
    free(h_odata);

    // free device memory
    CHECK(hipFree(d_idata));
    CHECK(hipFree(d_odata));

    // reset device
    CHECK(hipDeviceReset());

    // check the results
    bResult = (gpu_sum == cpu_sum);

    if(!bResult) printf("Test failed!\n");

    return EXIT_SUCCESS;
}
